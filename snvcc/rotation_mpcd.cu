#include "hip/hip_runtime.h"
#include "parameters.cuh"

__device__ double d_asin(double x) { return asin(x); }

__global__ void d_rotation_mpcd(point *vel_fl, point *pos_fl, int *fluid_no, int **cell_part, int no_of_fluid, 
						point len, double kbt, double mass_fl) {
	int k, cell_no;
	double r[4], ir[4], theta, phi, rho, var, scale_fac_mpcd, ct, st, ict;
	point *cell_vel, del_v, rr, rot[4], temp;
	cell_vel = (point *)malloc((len.prod() + 2)*sizeof(point));
	memset(fluid_no, 0, (len.prod() + 2)*sizeof(int));
	rr.random(point(0.5, 0.5, 0.5));
	for(int i = 1; i <= no_of_fluid; i++) {
		temp = mod(pos_fl[i] + rr, len);
		cell_no = 1 + temp.cell(len);
		cell_part[++fluid_no[cell_no]][cell_no] = i;
	}
	memset(cell_vel, 0, sizeof cell_vel);
	for(int i = 1; i <= len.prod(); i++) {
		if (fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++)
				cell_vel[i] += vel_fl[cell_part[j][i]]/fluid_no[i];
			rho = 2*ran() - 1, phi = 4.0*d_asin(1.0)*ran(), theta = 2*d_asin(1)*(130/180.0);
			r[1] = cos(phi)*sqrt(1 - rho*rho), r[2] = sin(phi)*sqrt(1 - rho*rho), r[3] = rho;
			ct = cos(theta), st = sin(theta), ict = 1 - cos(theta);
			ir[1] = ict*r[1], ir[2] = ict*r[2], ir[3] = ict*r[3];
			rot[1] = point(ir[1]*r[1] + ct, 	 ir[1]*r[2] - st*r[3], ir[1]*r[3] + st*r[2]);
			rot[2] = point(ir[2]*r[1] + st*r[3], ir[2]*r[2] + ct, 	   ir[2]*r[3] - st*r[1]);
			rot[3] = point(ir[3]*r[1] - st*r[2], ir[3]*r[2] + st*r[1], ir[3]*r[3] + ct);
			for(int j = 1;j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + point((rot[1]*del_v).sum(), (rot[2]*del_v).sum(), (rot[3]*del_v).sum());
			}
		}
	}
	for(int i = 1; i <= len.prod(); i++) {
		var = 0.0;
		if(fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				var += (del_v*del_v).sum();
			}
			scale_fac_mpcd = sqrt(3.0 * (fluid_no[i] - 1) * kbt/(mass_fl * var));
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + del_v*scale_fac_mpcd;
			}
		}
	}
}

void rotation_mpcd() {
	point *d_vel_fl, *d_pos_fl;
	int *d_fluid_no, **d_cell_part, *h_cell_part[128];
	hipMalloc(&d_vel_fl, (no_of_fluid + 2)*sizeof(point));
	hipMalloc(&d_pos_fl, (no_of_fluid + 2)*sizeof(point));
	hipMalloc(&d_fluid_no, (len.prod() + 2)*sizeof(int));
	hipMalloc(&d_cell_part, (maxpart + 2)*sizeof(int *));
	for(int i = 0; i <= maxpart; i++) {
		hipMalloc(&h_cell_part[i], sizeof(int)*(len.prod() + 2));
		hipMemcpy(h_cell_part[i], cell_part[i], sizeof(int)*(len.prod() + 2), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_cell_part, h_cell_part, (maxpart + 2)*sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(d_pos_fl, pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_vel_fl, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_fluid_no, fluid_no, (len.prod() + 2)*sizeof(point), hipMemcpyHostToDevice);
	d_rotation_mpcd<<<1, 1>>> (d_vel_fl, d_pos_fl, d_fluid_no, d_cell_part, no_of_fluid, len, kbt, mass_fl);
	hipMemcpy(pos_fl, d_pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipMemcpy(vel_fl, d_vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipMemcpy(fluid_no, d_fluid_no, (len.prod() + 2)*sizeof(point), hipMemcpyDeviceToHost);
	for(int i = 0; i <= maxpart; i++) {
		hipMemcpy(cell_part[i], h_cell_part[i], sizeof(int)*(len.prod() + 2), hipMemcpyDeviceToHost);
		hipFree(h_cell_part[i]);
	}
	hipFree(d_pos_fl), hipFree(d_vel_fl), hipFree(fluid_no), hipFree(d_cell_part);
}