#include "hip/hip_runtime.h"
#include "parameters.cuh"

__global__ void d_tumble(point *ra, point *pos_colloid, point len, int no_of_colloid){
	for (int i = 1; i <= no_of_colloid; i++) {
		ra[i] = img(pos_colloid[i] - ra[i].random(point(0, 0, 0), len), len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
    }
}

void tumble() {
	point *d_ra, *d_pos_colloid;
	hipMalloc(&d_ra, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_pos_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMemcpy(d_pos_colloid, pos_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_ra, ra, sizeof(point)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	d_tumble<<<1, 1>>>(d_ra, d_pos_colloid, len, no_of_colloid);
	hipMemcpy(pos_colloid, d_pos_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(ra, d_ra, sizeof(point)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipFree(d_ra), hipFree(d_pos_colloid);
}

__global__ void d_run(point *ra, point *vel_colloid, point *vel_fl, point *pos_fl, point *pos_colloid, point len, 
				int *no_neigh, int **nbr, int **neigh_fl, int *cnt, int *up_cnt, int no_of_colloid, double mass_fl, 
				double v0, double mass_colloid, double sigma) {
	point vector, del;
	double temp;
	for(int i = 1; i <= no_of_colloid; i++) {
		vel_colloid[i] += ra[i]*v0, del = ra[i]*v0;
		cnt[i] = up_cnt[i] = 0;
		for(int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[j][i]] - pos_colloid[i], len);
			if((vector*vector).sum() <= power(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[++cnt[i]][i] = neigh_fl[j][i];
 		}
		for(int j = 1; j <= cnt[i]; j++) {
			temp = mass_colloid/(mass_fl*cnt[i]);
			vel_fl[nbr[j][i]] = vel_fl[nbr[j][i]] - del*temp;
		}
	}
}

void run() {
	point *d_ra, *d_vel_colloid, *d_vel_fl, *d_pos_fl, *d_pos_colloid;
	int *d_no_neigh, **d_nbr, **d_neigh_fl, *d_cnt, *d_up_cnt, *h_nbr[7005], *h_neigh_fl[10005];
	hipMalloc(&d_ra, sizeof(point)*sizeof(no_of_colloid + 2));
	hipMalloc(&d_vel_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_pos_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_vel_fl, sizeof(point)*(no_of_fluid + 2));
	hipMalloc(&d_pos_fl, sizeof(point)*(no_of_fluid + 2));
	hipMalloc(&d_no_neigh, sizeof(int)*(no_of_colloid + 2));
	hipMalloc(&d_cnt, sizeof(int)*(no_of_colloid + 2));
	hipMalloc(&d_up_cnt, sizeof(int)*(no_of_colloid + 2));
	hipMalloc(&d_nbr, sizeof(int *)*7005);
	hipMalloc(&d_neigh_fl, sizeof(int *)*10005);

	hipMemcpy(d_ra, ra, sizeof(point)*sizeof(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_vel_colloid, vel_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_pos_colloid, pos_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_vel_fl, vel_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_pos_fl, pos_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_no_neigh, no_neigh, sizeof(int)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_cnt, cnt, sizeof(int)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_up_cnt, up_cnt, sizeof(int)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	for(int i = 0; i <= 10000; i++) {
		if(i <= 7000) {
			hipMalloc(&h_nbr[i], sizeof(int)*(no_of_colloid + 2));
			hipMemcpy(h_nbr[i], nbr[i], sizeof(int)*(no_of_colloid + 2), hipMemcpyHostToDevice);
		}
		hipMalloc(&h_neigh_fl[i], sizeof(int)*(no_of_colloid + 2));
		hipMemcpy(h_neigh_fl[i], neigh_fl[i], sizeof(int)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_nbr, h_nbr, sizeof(int *)*7005, hipMemcpyHostToDevice);
	hipMemcpy(d_neigh_fl, h_neigh_fl, sizeof(int *)*10005, hipMemcpyHostToDevice);
	d_run<<<1, 1>>>(d_ra, d_vel_colloid, d_vel_fl, d_pos_fl, d_pos_colloid, len, d_no_neigh, d_nbr, d_neigh_fl, 
		d_cnt, d_up_cnt, no_of_colloid, mass_fl, v0, mass_colloid, sigma);

	hipMemcpy(ra, d_ra, sizeof(point)*sizeof(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_colloid, d_vel_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(pos_colloid, d_pos_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_fl, d_vel_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(pos_fl, d_pos_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(no_neigh, d_no_neigh, sizeof(int)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(cnt, d_cnt, sizeof(int)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(up_cnt, d_up_cnt, sizeof(int)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	for(int i = 0; i <= 10000; i++) {
		if(i <= 7000) {
			hipMemcpy(nbr[i], h_nbr[i], sizeof(int)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
			hipFree(nbr[i]);
		}
		hipMemcpy(neigh_fl[i], h_neigh_fl[i], sizeof(int)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
		hipFree(neigh_fl[i]);
	}
	hipFree(d_ra), hipFree(d_pos_fl), hipFree(d_vel_fl), hipFree(d_cnt), hipFree(d_up_cnt), hipFree(d_nbr);
	hipFree(d_pos_colloid), hipFree(d_vel_colloid), hipFree(d_no_neigh), hipFree(d_neigh_fl);
}
void updown_velocity() {
	point up_vel = point(0, 0, 0), vector, vel;
	for (int i = 1; i <= no_of_colloid; i++){
		cnt[i] = 0, up_cnt[i] = 0, vel = point(0, 0, 0);
		for (int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[j][i]] - pos_colloid[i], len);
			
			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				nbr[++cnt[i]][i] = neigh_fl[j][i];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_nbr[++up_cnt[i]][i] = neigh_fl[j][i];
		}
		for (int j = 1; j <= cnt[i]; j++)
			vel += vel_fl[nbr[j][i]];

		for(int j = 1; j <= up_cnt[i]; j++)
			up_vel += vel_fl[up_nbr[j][i]];

		up_vel = (up_cnt[i] > 0)? up_vel/up_cnt[i] - vel_colloid[i]: up_vel;
		vel    = (up_cnt[i] > 0)? vel/cnt[i] - vel_colloid[i]: vel;
	}
}