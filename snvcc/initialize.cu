#include "hip/hip_runtime.h"
#include "parameters.cuh"

point *pos_colloid, *pos_fl, *vel_colloid, *vel_fl, *ang_vel_colloid, *f, *ra, *old_force, len = point(30, 30, 30);
int n = 10, niter = 21000, file = 0, nbin = 300, maxpart = 100, no_of_colloid = 10, nbox, **nbr, **up_nbr, *cnt, *up_cnt, *fluid_no;
int no_of_fluid = len.prod()*10, *no_neigh, *neigh_fl[10005], *neighbour[256], *n_neighbour, *box_neigh[512], **box_part, **cell_part, nn;
double kbt = 1, kbt1 = 1, ndt = 0.1, dv = 0.1, mass_fl = 1.0, mass_colloid = 654.1, sig_colloid = 5.0, eps = 1.0, v0 = 0.04;
double dt = ndt/(double)n, sigma = 0.80*sig_colloid, I_colloid = 0.1*mass_colloid*sigma*sigma, potential_colloid;

void initialize() {
	point **ppointers[]  = {&pos_fl, &vel_fl, &f, &pos_colloid, &vel_colloid, &ang_vel_colloid, &old_force, &ra};
	int   **ipointers[]  = {&fluid_no, &n_neighbour, &no_neigh, &cnt, &up_cnt};
	int isize[]          = {(int)len.prod(), no_of_colloid };
	int psize[]          = {no_of_fluid, no_of_colloid};

	box_part 	= (int **)calloc((maxpart + 2),sizeof(int *));
	cell_part 	= (int **)calloc((maxpart + 2),sizeof(int *));
	nbr 		= (int **)calloc(7005,sizeof(int *));
	up_nbr 		= (int **)calloc(7005,sizeof(int *));

	for(int i = 0; i < 8; i++) {
		if(i < 5) *ipointers[i] = (int   *)calloc(isize[i>0] + 2, sizeof(int)  );
				  *ppointers[i] = (point *)calloc(psize[i>1] + 2, sizeof(point));
	}
	for(int i = 0; i <= 10000; i++) {
		if(i <= 500)      box_neigh[i] = (int *)calloc(sizeof(int),(len.prod()    + 2));
		if(i <= maxpart)  box_part[i]  = (int *)calloc(sizeof(int),(len.prod()    + 2));
		if(i <= maxpart)  cell_part[i] = (int *)calloc(sizeof(int),(len.prod()    + 2));
		if(i <= 200)      neighbour[i] = (int *)calloc(sizeof(int),(no_of_colloid + 2));
		if(i <= 7000)     nbr[i]       = (int *)calloc(sizeof(int),(no_of_colloid + 2));
		if(i <= 7000)     up_nbr[i]    = (int *)calloc(sizeof(int),(no_of_colloid + 2));
						  neigh_fl[i]  = (int *)calloc(sizeof(int),(no_of_colloid + 2));
	}
}
__global__ void d_initialize_colloid(point *pos_colloid, point *vel_colloid, point *ang_vel_colloid, double sig_colloid, double kbt1, 
									double I_colloid, double mass_colloid, int no_of_colloid, point len) {
	int counter = 0, check, nofp = 0;
	double space_limit = 1.3*sig_colloid, ang_vscale_colloid = sqrt(12.0*kbt1/I_colloid), vscale_colloid = sqrt(12.0*kbt1/mass_colloid);
	point avr_vel = point(0, 0, 0), t, temp, iter = point(4, 4, 4), lim = len - point(1, 1, 1);

	for(int i = 0; i <= lim.prod(); i += 5, iter.next(lim, point(5, 5, 5), point(4, 4, 4)), nofp++) {
		if(nofp < no_of_colloid) pos_colloid[++nofp] = iter;
		else break;
	}
	while(counter < no_of_colloid) {
		t = t.random(point(0, 0, 0), len);
		check = 1;
		for(int j = 1; j <= counter; j++) {
			d_img(&temp, t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < space_limit)? 0: check;
        }
		if(check)
			pos_colloid[++counter] = t;
	}
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = vel_colloid[j].random(point(0.5, 0.5, 0.5))*vscale_colloid;
		avr_vel += vel_colloid[j];
	}
	avr_vel = avr_vel/no_of_colloid;
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = vel_colloid[j] - avr_vel;
		ang_vel_colloid[j] = (t.random(point(0.5, 0.5, 0.5)))*ang_vscale_colloid;
	}
}
void initialize_colloid() {
	point *d_pos_colloid, *d_vel_colloid, *d_ang_vel_colloid;
	hipMalloc(&d_pos_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_vel_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_ang_vel_colloid, sizeof(point)*(no_of_colloid + 2));
	d_initialize_colloid<<<1, 1>>> (d_pos_colloid, d_vel_colloid, d_ang_vel_colloid, sig_colloid, kbt1, 
							I_colloid, mass_colloid, no_of_colloid, len); 
	hipMemcpy(vel_colloid, d_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipMemcpy(ang_vel_colloid, d_ang_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipMemcpy(pos_colloid, d_pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_ang_vel_colloid), hipFree(d_vel_colloid), hipFree(d_pos_colloid);
}

__global__ void d_initialize_fluid(point *pos_fl, point *vel_fl, point *pos_colloid, double kbt, double mass_fl, 
								double sigma, int no_of_fluid, int no_of_colloid, point len) {
	int counter = 0, check;
	double vscale_fluid = sqrt(12.0*kbt/mass_fl);
	point avr_vel = point(0, 0, 0), t, temp;
	while(counter < no_of_fluid) {
		t = t.random(point(0, 0, 0), len);
		check = 1;
		for(int j = 1; j <= no_of_colloid; j++) {
			d_img(&temp, t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < sigma*0.5)? 0: check;
		}
		if(check)
			pos_fl[++counter] = t;
	}
	for(int j = 1; j <= no_of_fluid; j++) {
		vel_fl[j] = vel_fl[j].random(point(0.5, 0.5, 0.5))*vscale_fluid;
		avr_vel += vel_fl[j];
	}
	avr_vel = avr_vel/no_of_fluid;
	for(int j = 1; j <= no_of_fluid; j++)
		vel_fl[j] = vel_fl[j] - avr_vel;
}

void initialize_fluid() {
	point *d_pos_fl, *d_vel_fl, *d_pos_colloid;
	hipMalloc(&d_pos_fl, sizeof(point)*(no_of_fluid + 2));
	hipMalloc(&d_vel_fl, sizeof(point)*(no_of_fluid + 2));
	hipMalloc(&d_pos_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMemcpy(d_pos_colloid, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	d_initialize_fluid<<<1, 1>>> (d_pos_fl, d_vel_fl, d_pos_colloid, kbt, mass_fl, sigma, no_of_fluid, no_of_colloid, len); 
	hipMemcpy(vel_fl, d_vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipMemcpy(pos_fl, d_pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_pos_fl), hipFree(d_vel_fl), hipFree(d_pos_colloid);
}