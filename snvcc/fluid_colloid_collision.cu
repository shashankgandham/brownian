#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline __device__ point crossmul(point a, point b) {
	return point(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

inline __device__ point stochastic_reflection(point rf, point rs, double mass_fl, double kbt, point len) {
	double m_beta = mass_fl/kbt, random_e = power(1 - ran(), 2), val, v[4], x[4], z = 2;
	point un, ut, t, n;
	n = rs/sqrt((rs*rs).sum());
	val = sqrt(-log(random_e)/m_beta);

	un = n*val;
	t = img(t.random(rf, len), len);
	ut = crossmul(un, t);
	ut = ut/sqrt((ut*ut).sum());
	while(z > 1) {
		x[1] = 2.0 * ran() - 1, x[2] = 2.0 * ran() - 1;
		z = x[1]*x[1] + x[2]*x[2];
	}
	z = sqrt((-2.0*log(z))/z);
	v[1] = x[1]*z*sqrt(kbt/mass_fl); v[2] = x[2]*z*sqrt(kbt/mass_fl);
	return ut*v[1] + un;
}
__global__ void d_fluid_colloid_collision(int *no_neigh, point *pos_colloid, point *pos_fl, point *vel_colloid, 
										point *ang_vel_colloid, point *dump_vel_fl, double mass_colloid, point I_colloid,
										double mass_fl, double dt, point *vel_fl, point len, double sigma, int no_of_colloid,
										double kbt, int **neigh_fl) {
	point rr, rs, u, omega, vc;
//	int j = blockIdx.x, i = blockIdx.y;
	for(int j = 1; j <= no_of_colloid; j++) {
		vc = omega = point(0, 0, 0);
		for(int i = 1; i <= no_neigh[j]; i++) {
			int l = neigh_fl[i][j];
			rr = img(pos_colloid[j] - pos_fl[l], len);
			if((rr*rr).sum() <= pow(sigma, 2)*0.25) {
				pos_fl[l] = mod(pos_fl[l] - vel_fl[l]*dt* 0.5, len);
				rs = img(pos_fl[l] - pos_colloid[j], len);
				u  = stochastic_reflection(pos_fl[l], rs, mass_fl, kbt, len);
				vel_fl[l] = u + vel_colloid[j] + crossmul(ang_vel_colloid[j], rs);
				vc += (dump_vel_fl[l] - vel_fl[l]);
				u = (dump_vel_fl[l] - vel_fl[l]);
				omega += crossmul(rs, (dump_vel_fl[l] - vel_fl[l]));
				pos_fl[l] = mod(pos_fl[l] + vel_fl[l]*dt*0.5, len);
			}
		}
		vel_colloid[j] 	   += vc*mass_fl/mass_colloid;
		ang_vel_colloid[j] += omega*mass_fl/I_colloid;
	}
}

void fluid_colloid_collision() {
	point *dump_vel_fl, *d_pos_colloid, *d_pos_fl, *d_vel_colloid, *d_ang_vel_colloid, *d_vel_fl;
	int *d_no_neigh, **d_neigh_fl, *h_neigh_fl[10000];
	hipMalloc(&dump_vel_fl, sizeof(point)*(no_of_fluid + 2));
	hipMalloc(&d_vel_fl, sizeof(point)*(no_of_fluid + 2));
	hipMalloc(&d_pos_fl, sizeof(point)*(no_of_fluid + 2));
	hipMalloc(&d_pos_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_vel_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_ang_vel_colloid, sizeof(point)*(no_of_colloid + 2));
	hipMalloc(&d_no_neigh, sizeof(int)*(no_of_colloid));
	hipMalloc(&d_neigh_fl, 10000*sizeof(int *));
	for(int i = 0; i <= 200; i++) {
		hipMalloc(&h_neigh_fl[i], (no_of_colloid + 2)*sizeof(int));
		hipMemcpy(h_neigh_fl[i], neigh_fl[i], (no_of_colloid + 2)*sizeof(int), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_neigh_fl, h_neigh_fl, 10000*sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(dump_vel_fl, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_vel_fl, vel_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_pos_fl, pos_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_pos_colloid, pos_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_vel_colloid, vel_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_ang_vel_colloid, ang_vel_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyHostToDevice);
	hipMemcpy(d_no_neigh, no_neigh, sizeof(int)*(no_of_colloid), hipMemcpyHostToDevice);
	d_fluid_colloid_collision<<<1, 1>>> (d_no_neigh, d_pos_colloid, d_pos_fl, d_vel_colloid, d_ang_vel_colloid, dump_vel_fl, 
	mass_colloid, I_colloid, mass_fl, dt, d_vel_fl, len, sigma, no_of_colloid, kbt, d_neigh_fl);
	hipMemcpy(vel_fl, d_vel_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(pos_fl, d_pos_fl, sizeof(point)*(no_of_fluid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(pos_colloid, d_pos_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(vel_colloid, d_vel_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipMemcpy(ang_vel_colloid, d_ang_vel_colloid, sizeof(point)*(no_of_colloid + 2), hipMemcpyDeviceToHost);
	hipFree(dump_vel_fl), hipFree(d_vel_fl), hipFree(d_pos_fl), hipFree(d_pos_colloid);
	hipFree(d_vel_colloid), hipFree(d_ang_vel_colloid), hipFree(d_no_neigh), hipFree(d_neigh_fl);
	for(int i = 0; i <= 200; i++) 
		hipFree(h_neigh_fl[i]);
}