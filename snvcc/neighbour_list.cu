#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline point cmod(point a, point b) {
    if(a.x <=  0) a.x += b.x;  if(a.y <=  0) a.y += b.y;
    if(a.z <=  0) a.z += b.z; if(a.x > b.x) a.x -= b.x;
    if(a.y > b.y) a.y -= b.y; if(a.z > b.z) a.z -= b.z;
    return a;
}
void create_box() {
    int tbox, box;
    point jiter, iter = point(1, 1, 1), temp;
    for(int i = 1; i <= len.prod(); i++, iter.next(len)) {
        nbox = 0, box = (iter - point(0, 1, 1)).cell(len);
        jiter = iter - point(3, 3, 3);
        for(int j = 1; j <= 343; j++) {
            tbox = (cmod(jiter, len) - point(0, 1, 1)).cell(len);
            if(tbox != box) box_neigh[++nbox][box] = tbox;
            jiter.next(iter + point(3, 3, 3), point(1, 1, 1), iter - point(3, 3, 3));
        }
    }
}
__global__ void d_neighbour_list_md(int **neighbour, int *n_neighbour, point *pos_colloid, int no_of_colloid, double sig_colloid, point len) {
    double neigh_cutoff = 3.0*sig_colloid;
    point temp;
    for(int i = 1; i < no_of_colloid; i++) {
        for(int j = i + 1; j <= no_of_colloid; j++) {
            d_img(&temp, pos_colloid[i] - pos_colloid[j], len);  
            if((temp*temp).sum() < power(neigh_cutoff,2)) 
                neighbour[++n_neighbour[i]][i] = j;
        }
    }
}
__global__ void d_neighbour_list_mpcd(int **box_part, int *fluid_no, int **box_neigh, int **neigh_fl, int *no_neigh,
                    point *pos_colloid, point *pos_fl, int no_of_fluid, int no_of_colloid, int nbox, point len) {
    int box_no, mm, cbox;
    for(int	i = 1; i <= no_of_fluid; i++) {
        box_no = 1 + pos_fl[i].cell(len);
        box_part[++fluid_no[box_no]][box_no] = i;
    }
    for(int j = 1; j <= no_of_colloid; j++) {
        no_neigh[j] = 0;
        cbox = 1 + pos_colloid[j].cell(len);
        for(int k = 1; k <= nbox; k++) {
            mm = box_neigh[k][cbox];
            for(int i = 1; i <= fluid_no[mm]; i++) {
                neigh_fl[++no_neigh[j]][j] = box_part[i][mm];
            }
        }
    }
}

void neighbour_list_mpcd() {
    memset(fluid_no, 0, sizeof(int)*(len.prod() + 2));
    point *d_pos_colloid, *d_pos_fl;
    int *d_fluid_no, **d_box_part, **d_neigh_fl, **d_box_neigh, *d_no_neigh;
    int *h_neigh_fl[10005], *h_box_part[128], *h_box_neigh[512];
	hipMalloc(&d_pos_colloid, (no_of_colloid + 2)*sizeof(point)); 
    hipMalloc(&d_pos_fl, (no_of_fluid + 2)*sizeof(point));
    hipMalloc(&d_fluid_no, (no_of_fluid + 2)*sizeof(point));
    hipMalloc(&d_no_neigh, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_neigh_fl, (10005)*sizeof(int *));
    hipMalloc(&d_box_part, (maxpart + 2)*sizeof(int *));
    hipMalloc(&d_box_neigh, (512)*sizeof(int *));
	for(int i = 0; i <= 10000; i++) {
		hipMalloc(&h_neigh_fl[i], (no_of_colloid + 2)*sizeof(int));
        hipMemcpy(h_neigh_fl[i], neigh_fl[i], (no_of_colloid + 2)*sizeof(int), hipMemcpyHostToDevice);
        if(i <= maxpart) {
            hipMalloc(&h_box_part[i], (len.prod() + 2)*sizeof(int));
            hipMemcpy(h_box_part[i], box_part[i], (len.prod() + 2)*sizeof(int), hipMemcpyHostToDevice);
        }
        if(i <= 500) {
            hipMalloc(&h_box_neigh[i], (len.prod() + 2)*sizeof(int));
            hipMemcpy(h_box_neigh[i], box_neigh[i], (len.prod() + 2)*sizeof(int), hipMemcpyHostToDevice);
        }
    }
    hipMemcpy(d_pos_colloid, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice ); 
    hipMemcpy(d_pos_fl, pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
    hipMemcpy(d_fluid_no, fluid_no, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
    hipMemcpy(d_no_neigh, no_neigh, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
    hipMemcpy(d_neigh_fl, h_neigh_fl, (10005)*sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(d_box_part, h_box_part, (maxpart + 2)*sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(d_box_neigh, h_box_neigh, (512)*sizeof(int *), hipMemcpyHostToDevice);
    d_neighbour_list_mpcd<<<1, 1>>>(d_box_part, d_fluid_no, d_box_neigh, d_neigh_fl, d_no_neigh, d_pos_colloid, d_pos_fl, 
                            no_of_fluid, no_of_colloid, nbox, len);    
   for(int i = 0; i <= 10000; i++) {
        hipMemcpy(neigh_fl[i], h_neigh_fl[i], (no_of_colloid + 2)*sizeof(int), hipMemcpyDeviceToHost);
        hipFree(neigh_fl[i]);
        if(i <= 500) hipFree(h_box_neigh[i]);
        if(i <= maxpart) {    
            hipMemcpy(box_part[i], h_box_part[i], (len.prod() + 2)*sizeof(int), hipMemcpyDeviceToHost);
            hipFree(h_box_part[i]);
        }
    }
    hipMemcpy(fluid_no, d_fluid_no, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
    hipMemcpy(no_neigh, d_no_neigh, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
    hipFree(d_fluid_no), hipFree(d_no_neigh), hipFree(d_pos_colloid), hipFree(d_pos_fl);
    hipFree(d_box_part), hipFree(d_box_neigh), hipFree(d_neigh_fl);
}

void neighbour_list_md() {
    memset(n_neighbour, 0, sizeof(int)*(no_of_colloid + 2));
    point *d_pos;
	int *d_n_neighbour, **d_neighbour, *h_neighbour[256];
	hipMalloc(&d_n_neighbour, (no_of_colloid + 2)*sizeof(int)); 
    hipMalloc(&d_neighbour, 256*sizeof(int *));
    hipMalloc(&d_pos, sizeof(point)*(no_of_colloid + 2));
	for(int i = 0; i <= 200; i++) {
		hipMalloc(&h_neighbour[i], (no_of_colloid + 2)*sizeof(int));
		hipMemcpy(h_neighbour[i], neighbour[i], (no_of_colloid + 2)*sizeof(int), hipMemcpyHostToDevice);
    }
    hipMemcpy(d_neighbour, h_neighbour, 256*sizeof(int *), hipMemcpyHostToDevice);
    hipMemcpy(d_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
    hipMemcpy(d_n_neighbour, n_neighbour, (no_of_colloid + 2)*sizeof(int), hipMemcpyHostToDevice);  
  	d_neighbour_list_md<<<1, 1>>>(d_neighbour, d_n_neighbour, d_pos, no_of_colloid, sig_colloid, len);
    for(int i = 0; i <= 200; i++) {
		hipMemcpy(neighbour[i], h_neighbour[i], (no_of_colloid + 2)*sizeof(int), hipMemcpyDeviceToHost);
        hipFree(h_neighbour[i]);
    }
    hipFree(d_neighbour), hipFree(d_n_neighbour), hipFree(d_pos);
}