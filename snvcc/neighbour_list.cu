#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline point cmod(point a, point b) {
    if(a.x <=  0) a.x += b.x;  if(a.y <=  0) a.y += b.y;
    if(a.z <=  0) a.z += b.z; if(a.x > b.x) a.x -= b.x;
    if(a.y > b.y) a.y -= b.y; if(a.z > b.z) a.z -= b.z;
    return a;
}
void create_box() {
    int tbox, box;
    point jiter, iter = point(1, 1, 1), temp;
    for(int i = 1; i <= len.prod(); i++, iter.next(len)) {
        nbox = 0, box = (iter - point(0, 1, 1)).cell(len);
        jiter = iter - point(3, 3, 3);
        for(int j = 1; j <= 343; j++) {
            tbox = (cmod(jiter, len) - point(0, 1, 1)).cell(len);
            if(tbox != box) box_neigh[++nbox][box] = tbox;
            jiter.next(iter + point(3, 3, 3), point(1, 1, 1), iter - point(3, 3, 3));
        }
    }
}
__global__ void d_neighbour_list_md(int **neighbour, int *n_neighbour, point *pos_colloid, int no_of_colloid, double sig_colloid, point len) {
    double neigh_cutoff = 3.0*sig_colloid;
    point temp;
    memset(n_neighbour, 0, sizeof(int)*(no_of_colloid + 2));
    int i = blockIdx.x + 1, j = blockIdx.x + blockIdx.y + 1;
    if(j <= no_of_colloid) {
      d_img(&temp, pos_colloid[i] - pos_colloid[j], len);  
      if((temp*temp).sum() < pow(neigh_cutoff,2)) 
        neighbour[++n_neighbour[i]][i] = j;
    }
}
void neighbour_list_md() {
    point *d_pos;
    memset(n_neighbour, 0, sizeof(int)*(no_of_colloid + 2));
	int *d_n_neighbour, **d_neighbour, *h_neighbour[256];
	hipMalloc(&d_n_neighbour, (no_of_colloid + 2)*sizeof(int)); 
    hipMalloc(&d_neighbour, 256*sizeof(int *));
    hipMalloc(&d_pos, sizeof(point)*(no_of_colloid + 2));
	for(int i = 0; i <= 200; i++) {
		hipMalloc(&h_neighbour[i], (no_of_colloid + 2)*sizeof(int));
		hipMemcpy(h_neighbour[i], neighbour[i], (no_of_colloid + 2)*sizeof(int), hipMemcpyHostToDevice);
	}
  	hipMemcpy(d_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	d_neighbour_list_md<<<no_of_colloid + 1, 1>>>(d_neighbour, d_n_neighbour, d_pos, no_of_colloid, sig_colloid, len);
  	hipMemcpy(neighbour, d_neighbour, 256*(no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
}

void neighbour_list_mpcd() {
    int box_no, mm, cbox;
    memset(fluid_no, 0, sizeof(int)*(len.prod() + 2));
    for(int	i = 1; i <= no_of_fluid; i++) {
        box_no = 1 + pos_fl[i].cell(len);
        box_part[++fluid_no[box_no]][box_no] = i;
    }
    for(int j = 1; j <= no_of_colloid; j++) {
        no_neigh[j] = 0;
        cbox = 1 + pos_colloid[j].cell(len);
        for(int k = 1; k <= nbox; k++) {
            mm = box_neigh[k][cbox];
            for(int i = 1; i <= fluid_no[mm]; i++) {
                neigh_fl[++no_neigh[j]][j] = box_part[i][mm];
            }
        }
    }
}