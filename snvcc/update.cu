#include "hip/hip_runtime.h"
#include "parameters.cuh"

double mag_f, r_cutoff = pow(2, 1.0/6.0)*sig_colloid, r;
double fc = 4.0*eps*(12.0*(pow(sig_colloid,12)/pow(r_cutoff,13)) - 6.0*(pow(sig_colloid, 6)/pow(r_cutoff, 7)));
double ufc = 4.0*eps*(pow(sig_colloid/r_cutoff, 12) - pow(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;
double sig_colloid12 = pow(sig_colloid, 12), sig_colloid6 = pow(sig_colloid, 6);
double dtb2 = dt/(mass_colloid*2);

double power(double x, int r) {
    double ans = 1;
    for(int i = 1; i <=r; i++)
        ans *= x;
    return ans;
}
void compute_force_md() {
	point temp, ff;
	potential_colloid = 0;
    memset(f, 0, sizeof(int)*(no_of_colloid + 2));
    double t1, t2;
	for(int i = 1; i <= no_of_colloid; i++) {
		for(int j = 1; j <= n_neighbour[i]; j++) {
			temp = img(pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
			r = sqrt((temp*temp).sum());
			if(r < r_cutoff) {
				potential_colloid += 4*eps*(pow(sig_colloid/r, 12) - pow(sig_colloid/r, 6)) - ufc + fc*r;
				t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
                mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
                ff = (temp*mag_f)/r;
				f[i] += ff, f[neighbour[j][i]] -= ff;
            }
		}   
	}
}
//PARALLELIZED
__global__ void d_update_activity_direction(point *ang_vel_colloid, point *ra, double dt) {
	point m[4], b, sb, cb;
	b  = ang_vel_colloid[blockIdx.x]*dt;
	sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));
	m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
	m[2] =  point(sb.x*sb.y*cb.z + cb.x*sb.z, -sb.x*sb.y*sb.z + cb.x*cb.z, -sb.x*cb.y);
	m[3] =  point(-cb.x*sb.y*cb.z + sb.x*sb.z, cb.x*sb.y*sb.z + sb.x*cb.z, cb.x*cb.y);
	ra[blockIdx.x] = point((m[1]*ra[blockIdx.x]).sum(), (m[2]*ra[blockIdx.x]).sum(), (m[3]*ra[blockIdx.x]).sum());
}
__global__ void d_update_pos_md(point *pos_colloid, point *vel_colloid, point *f, double dt, double mass_colloid, point len) {
	double dt2= dt*dt, ddt = 0.5*dt2/mass_colloid;
	d_mod(&pos_colloid[blockIdx.x], pos_colloid[blockIdx.x] + vel_colloid[blockIdx.x]*dt + f[blockIdx.x]*ddt, len);
}
__global__ void d_update_pos_mpcd(point *pos_fl, point *vel_fl, double dt, point len) {
  	d_mod(&pos_fl[blockIdx.x], pos_fl[blockIdx.x] + vel_fl[blockIdx.x]*dt, len);
}
__global__ void d_update_vel_colloid(point *d_vel, point *d_old_force, point *d_f, double dtb2){
	d_vel[blockIdx.x] += d_old_force[blockIdx.x] + d_f[blockIdx.x]*dtb2; 
}

void update_activity_direction() {
	point *d_ang_vel, *d_ra;
    hipMalloc(&d_ang_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_ra,   	   (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(d_ang_vel, ang_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_ra, 	  ra			 , (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	d_update_activity_direction<<<no_of_colloid + 1, 1>>>(d_ang_vel, d_ra, dt);
  	hipMemcpy(ra, d_ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_ang_vel), hipFree(d_ra);	
}
void update_pos_md() {
	point *d_vel, *d_pos, *d_f;

    hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_pos, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_f,   (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(d_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_f, 			f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	d_update_pos_md<<<no_of_colloid + 1, 1>>>(d_pos, d_vel, d_f, dt, mass_colloid, len);
  	hipMemcpy(pos_colloid, d_pos, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_pos), hipFree(d_vel), hipFree(d_f);
	for(int i = 1; i <=10; i++)
		pos_colloid[i].print();
	printf("\n");
	exit(0);

}

void update_pos_mpcd() {
	point *d_pos, *d_vel;
    hipMalloc(&d_vel, (no_of_fluid + 2)*sizeof(point));
    hipMalloc(&d_pos, (no_of_fluid + 2)*sizeof(point)); 
  	hipMemcpy(d_vel, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_pos, pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	d_update_pos_mpcd<<<no_of_fluid + 1,1>>>(d_pos, d_vel, dt, len);
  	hipMemcpy(vel_fl, d_vel, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_vel), hipFree(d_pos);
}

void update_velocity_colloid() {
    point *d_vel, *d_old_force, *d_f;
    hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_old_force, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_f, (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(d_old_force, old_force, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	d_update_vel_colloid<<<no_of_colloid + 1, 1>>>(d_vel, d_old_force, d_f, dtb2);
  	hipMemcpy(vel_colloid, d_vel, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_old_force), hipFree(d_vel), hipFree(d_f);
}
