#include "hip/hip_runtime.h"
#include "parameters.cuh"

double dtb2 = dt/(mass_colloid*2);

__global__ void d_compute_force_md(point *f, int *n_neighbour, int *neighbour[], point *pos_colloid, double sig_colloid, double eps, point *potential_colloid, point len, int no_of_colloid) {
	double mag_f, r_cutoff, fc, ufc, sig_colloid12, sig_colloid6, r;
	r_cutoff = power(2, 1.0/6.0)*sig_colloid, r, fc = 4.0*eps*(12.0*(power(sig_colloid,12)/power(r_cutoff,13)) - 6.0*(power(sig_colloid, 6)/power(r_cutoff, 7)));
	ufc = 4.0*eps*(power(sig_colloid/r_cutoff, 12) - power(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;
	sig_colloid12 = power(sig_colloid, 12), sig_colloid6 = power(sig_colloid, 6);
	point temp, ff;
	double t1, t2;
	for(int i = 1; i <= no_of_colloid; i++) {
		for(int j = 1; j <= n_neighbour[i]; j++) {
			temp = img(pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
			r = sqrt((temp*temp).sum());
			if(r < r_cutoff) {
				*potential_colloid += 4*eps*(power(sig_colloid/r, 12) - power(sig_colloid/r, 6)) - ufc + fc*r;
				t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
				mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
				ff = (temp*mag_f)/r;
				f[i] += ff, f[neighbour[j][i]] -= ff;
			}   
		}
	}
}
__global__ void d_update_activity_direction(point *ang_vel_colloid, point *ra, double dt, int no_of_colloid) {
	point m[4], b, sb, cb;
	for(int i = 1; i <= no_of_colloid; i++) {
		b  = ang_vel_colloid[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));
		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		m[2] =  point((sb.x*sb.y)*cb.z + cb.x*sb.z, (-sb.x*sb.y)*sb.z + cb.x*cb.z, -sb.x*cb.y);
		m[3] =  point((-cb.x*sb.y)*cb.z + sb.x*sb.z, (cb.x*sb.y)*sb.z + sb.x*cb.z, cb.x*cb.y);
		ra[i] = point((m[1]*ra[i]).sum(), (m[2]*ra[i]).sum(), (m[3]*ra[i]).sum());
	}
}
__global__ void d_update_pos_md(point *pos_colloid, point *vel_colloid, point *f, double dt, double mass_colloid, point len, int no_of_colloid) {
	double dt2= dt*dt, ddt = 0.5*dt2/mass_colloid;
	for(int i = 1; i <= no_of_colloid; i++)
		pos_colloid[i] = mod(pos_colloid[i] + vel_colloid[i]*dt + f[i]*ddt, len);
}
__global__ void d_update_pos_mpcd(point *pos_fl, point *vel_fl, double dt, point len, int no_of_fluid) {
	for(int i = 1; i <= no_of_fluid; i++)
	pos_fl[i] = mod(pos_fl[i] + vel_fl[i]*dt, len);
}
__global__ void d_update_vel_colloid(point *d_vel, point *d_old_force, point *d_f, double dtb2, int no_of_colloid){
	for(int i = 1; i <= no_of_colloid; i++) 
		d_vel[i] += d_old_force[i] + d_f[i]*dtb2; 
}

void compute_force_md() {
	point *d_pos_colloid, *d_f, *d_potential_colloid;
	int *d_n_neighbour, **d_neighbour, *h_neighbour[256];
	potential_colloid = 0;
	memset(f, 0, sizeof(int)*(no_of_colloid + 2));
	hipMalloc(&d_potential_colloid, sizeof(point));
	hipMalloc(&d_pos_colloid, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_f, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_n_neighbour, (no_of_colloid + 2)*sizeof(int)); 
	hipMalloc(&d_neighbour, 256*sizeof(int *));
	for(int i = 0; i <= 200; i++) {
		hipMalloc(&h_neighbour[i], (no_of_colloid + 2)*sizeof(int));
		hipMemcpy(h_neighbour[i], neighbour[i], (no_of_colloid + 2)*sizeof(int), hipMemcpyHostToDevice);
	}
	hipMemcpy(d_neighbour, h_neighbour, 256*sizeof(int *), hipMemcpyHostToDevice);
	hipMemcpy(d_potential_colloid, &potential_colloid, sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_n_neighbour, n_neighbour, (no_of_colloid + 2)*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_pos_colloid, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
	d_compute_force_md<<<1, 1>>>(d_f, d_n_neighbour, d_neighbour, d_pos_colloid, sig_colloid, eps, d_potential_colloid, len, no_of_colloid);
	hipMemcpy(f, d_f, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_f), hipFree(d_pos_colloid), hipFree(d_n_neighbour), hipFree(d_potential_colloid);
	for(int i = 0; i <= 200; i++) hipFree(h_neighbour[i]);
	hipFree(d_neighbour);
}

void update_activity_direction() {
	point *d_ang_vel, *d_ra;
	hipMalloc(&d_ang_vel, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_ra, (no_of_colloid + 2)*sizeof(point)); 
	hipMemcpy(d_ang_vel, ang_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_ra, ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_activity_direction<<<1, 1>>>(d_ang_vel, d_ra, dt, no_of_colloid);
	hipMemcpy(ra, d_ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_ang_vel), hipFree(d_ra);
}
void update_pos_md() {
	point *d_vel, *d_pos, *d_f;

	hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_pos, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_f,   (no_of_colloid + 2)*sizeof(point)); 
	hipMemcpy(d_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_pos_md<<<1, 1>>>(d_pos, d_vel, d_f, dt, mass_colloid, len, no_of_colloid);
	hipMemcpy(pos_colloid, d_pos, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_pos), hipFree(d_vel), hipFree(d_f);
}

void update_pos_mpcd() {
	point *d_pos, *d_vel;
	hipMalloc(&d_vel, (no_of_fluid + 2)*sizeof(point));
	hipMalloc(&d_pos, (no_of_fluid + 2)*sizeof(point)); 
	hipMemcpy(d_vel, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_pos, pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_pos_mpcd<<<1, 1>>>(d_pos, d_vel, dt, len, no_of_fluid);
	hipMemcpy(pos_fl, d_pos, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_vel), hipFree(d_pos);
}

void update_velocity_colloid() {
	point *d_vel, *d_old_force, *d_f;
	hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_old_force, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_f, (no_of_colloid + 2)*sizeof(point)); 
	hipMemcpy(d_old_force, old_force, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_vel_colloid<<<1, 1>>>(d_vel, d_old_force, d_f, dtb2, no_of_colloid);
	hipMemcpy(vel_colloid, d_vel, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_old_force), hipFree(d_vel), hipFree(d_f);
}