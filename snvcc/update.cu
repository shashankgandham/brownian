#include "hip/hip_runtime.h"
#include "parameters.cuh"

double dtb2 = dt/(mass_colloid*2);

__device__ double power(double x, int r) {
	double ans = 1;
	for(int i = 1; i <=r; i++)
		ans *= x;
	return ans;
}

//PARALLELIZED
__global__ void d_compute_force_md(point *f, int *n_neighbour, int *neighbour[], int *pos_colloid, double sig_colloid, double eps, double *potential_colloid, point len) {
	double mag_f, r_cutoff, fc, ufc, sig_colloid12, sig_colloid6, r;
	r_cutoff = power(2, 1.0/6.0)*sig_colloid, r, fc = 4.0*eps*(12.0*(power(sig_colloid,12)/power(r_cutoff,13)) - 6.0*(power(sig_colloid, 6)/power(r_cutoff, 7)));
	ufc = 4.0*eps*(power(sig_colloid/r_cutoff, 12) - power(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;
	sig_colloid12 = power(sig_colloid, 12), sig_colloid6 = power(sig_colloid, 6);
	point temp, ff;
	double t1, t2;
	int i = blockIdx.x + 1, j = blockIdx.y + 1;
	if(j <= n_neighbour[i]) {
		d_img(&temp, pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
		r = sqrt((temp*temp).sum());
		if(r < r_cutoff) {
			potential_colloid += 4*eps*(power(sig_colloid/r, 12) - power(sig_colloid/r, 6)) - ufc + fc*r;
			t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
			mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
			ff = (temp*mag_f)/r;
			f[i] += ff, f[neighbour[j][i]] -= ff;
		}   
	}

}
__global__ void d_update_activity_direction(point *ang_vel_colloid, point *ra, double dt) {
	point m[4], b, sb, cb;
	int i = blockIdx.x + 1;
	b  = ang_vel_colloid[i]*dt;
	sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));
	m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
	m[2] =  point((sb.x*sb.y)*cb.z + cb.x*sb.z, (-sb.x*sb.y)*sb.z + cb.x*cb.z, -sb.x*cb.y);
	m[3] =  point((-cb.x*sb.y)*cb.z + sb.x*sb.z, (cb.x*sb.y)*sb.z + sb.x*cb.z, cb.x*cb.y);

	ra[i] = point((m[1]*ra[i]).sum(), (m[2]*ra[i]).sum(), (m[3]*ra[i]).sum());
}
__global__ void d_update_pos_md(point *pos_colloid, point *vel_colloid, point *f, double dt, double mass_colloid, point len) {
	double dt2= dt*dt, ddt = 0.5*dt2/mass_colloid;
	int i = blockIdx.x + 1;
	d_mod(&pos_colloid[i], pos_colloid[i] + vel_colloid[i]*dt + f[i]*ddt, len);
}
__global__ void d_update_pos_mpcd(point *pos_fl, point *vel_fl, double dt, point len) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x + 1;
	d_mod(&pos_fl[i], pos_fl[i] + vel_fl[i]*dt, len);
}
__global__ void d_update_vel_colloid(point *d_vel, point *d_old_force, point *d_f, double dtb2){
	int i = blockIdx.x + 1;
	d_vel[i] += d_old_force[i] + d_f[i]*dtb2; 
}

void compute_force_md() {
	point *d_pos_colloid; size_t Pitch;
	int *d_n_neighbour, **d_neighbour;
	potential_colloid = 0;
	memset(f, 0, sizeof(int)*(no_of_colloid + 2));
	hipMalloc(&d_pos_colloid, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_n_neighbour, (no_of_colloid + 2)*sizeof(int)); 
	hipMallocPitch(&d_neighbour, &Pitch, (no_of_colloid + 2)*sizeof(int)); 
	hipMemcpy(d_ang_vel, ang_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_ra, ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
	d_compute_force_md<<<no_of_colloid, 1>>>(d_f, d_n_neighbour, d_neighbour, d_pos_colloid, sig_colloid, eps, d_potential_colloid, len);
	hipMemcpy(ra, d_ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_ang_vel), hipFree(d_ra);

}

void update_activity_direction() {
	point *d_ang_vel, *d_ra;
	hipMalloc(&d_ang_vel, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_ra, (no_of_colloid + 2)*sizeof(point)); 
	hipMemcpy(d_ang_vel, ang_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_ra, ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_activity_direction<<<no_of_colloid, 1>>>(d_ang_vel, d_ra, dt);
	hipMemcpy(ra, d_ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_ang_vel), hipFree(d_ra);
}
void update_pos_md() {
	point *d_vel, *d_pos, *d_f;

	hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_pos, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_f,   (no_of_colloid + 2)*sizeof(point)); 
	hipMemcpy(d_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_pos_md<<<no_of_colloid, 1>>>(d_pos, d_vel, d_f, dt, mass_colloid, len);
	hipMemcpy(pos_colloid, d_pos, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_pos), hipFree(d_vel), hipFree(d_f);
}

void update_pos_mpcd() {
	point *d_pos, *d_vel;
	hipMalloc(&d_vel, (no_of_fluid + 2)*sizeof(point));
	hipMalloc(&d_pos, (no_of_fluid + 2)*sizeof(point)); 
	hipMemcpy(d_vel, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_pos, pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_pos_mpcd<<<no_of_fluid/1000, 1000>>>(d_pos, d_vel, dt, len);
	hipMemcpy(pos_fl, d_pos, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_vel), hipFree(d_pos);
}

void update_velocity_colloid() {
	point *d_vel, *d_old_force, *d_f;
	hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_old_force, (no_of_colloid + 2)*sizeof(point));
	hipMalloc(&d_f, (no_of_colloid + 2)*sizeof(point)); 
	hipMemcpy(d_old_force, old_force, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	

	d_update_vel_colloid<<<no_of_colloid, 1>>>(d_vel, d_old_force, d_f, dtb2);
	hipMemcpy(vel_colloid, d_vel, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
	hipFree(d_old_force), hipFree(d_vel), hipFree(d_f);
}
