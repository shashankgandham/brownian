#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline point cmod(point a, point b) {
    if(a.x <=  0) a.x += b.x;  if(a.y <=  0) a.y += b.y;
    if(a.z <=  0) a.z += b.z; if(a.x > b.x) a.x -= b.x;
    if(a.y > b.y) a.y -= b.y; if(a.z > b.z) a.z -= b.z;
    return a;
}
void create_box() {
    int tbox, box;
    point jiter, iter = point(1, 1, 1), temp;
    for(int i = 1; i <= len.prod(); i++, iter.next(len)) {
        nbox = 0, box = (iter - point(0, 1, 1)).cell(len);
        jiter = iter - point(3, 3, 3);
        for(int j = 1; j <= 343; j++) {
            tbox = (cmod(jiter, len) - point(0, 1, 1)).cell(len);
            if(tbox != box) box_neigh[++nbox][box] = tbox;
            jiter.next(iter + point(3, 3, 3), point(1, 1, 1), iter - point(3, 3, 3));
        }
    }
}
__global__ void d_neighbour_list_md(int **neighbour, int *n_neighbour, point *pos_colloid, int no_of_colloid, double sig_colloid, point len) {
    double neigh_cutoff = 3.0*sig_colloid;
    point temp;
    for(int i = 1; i < no_of_colloid; i++) {
        for(int j = i + 1; j <= no_of_colloid; j++) {
            temp = img(pos_colloid[i] - pos_colloid[j], len);  
            if((temp*temp).sum() < power(neigh_cutoff,2)) 
                neighbour[++n_neighbour[i]][i] = j;
        }
    }
}
__global__ void d_neighbour_list_mpcd(int **box_part, int *fluid_no, int **box_neigh, int **neigh_fl, int *no_neigh,
                    point *pos_colloid, point *pos_fl, int no_of_fluid, int no_of_colloid, int nbox, point len) {
    int box_no, mm, cbox;
    for(int	i = 1; i <= no_of_fluid; i++) {
        box_no = 1 + pos_fl[i].cell(len);
        box_part[++fluid_no[box_no]][box_no] = i;
    }
    for(int j = 1; j <= no_of_colloid; j++) {
        no_neigh[j] = 0;
        cbox = 1 + pos_colloid[j].cell(len);
        for(int k = 1; k <= nbox; k++) {
            mm = box_neigh[k][cbox];
            for(int i = 1; i <= fluid_no[mm]; i++) {
                neigh_fl[++no_neigh[j]][j] = box_part[i][mm];
            }
        }
    }
}

void neighbour_list_md() {
    hipMemset(n_neighbour, 0, sizeof(int)*(no_of_colloid + 2));  
    d_neighbour_list_md<<<1, 1>>>(neighbour, n_neighbour, pos_colloid, no_of_colloid, sig_colloid, len);
}

void neighbour_list_mpcd() {
    hipMemset(fluid_no, 0, sizeof(int)*(len.prod() + 2));
    d_neighbour_list_mpcd<<<1, 1>>>(box_part, fluid_no, box_neigh, neigh_fl, no_neigh, pos_colloid, pos_fl, no_of_fluid, no_of_colloid, nbox, len);
}
