#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

inline __device__ __host__ point cmod(point a, point b) {
	if(a.x <=  0) a.x += b.x;  if(a.y <=  0) a.y += b.y;
	if(a.z <=  0) a.z += b.z; if(a.x > b.x) a.x -= b.x;
	if(a.y > b.y) a.y -= b.y; if(a.z > b.z) a.z -= b.z;
	return a;
}

inline __device__ point xyz(int cell, point len) {
	int x = len.x, y = len.y, px, py, pz;
	px = (cell%(x*y))%x;
	if(!px) px = 30; cell -= px;
	py = (cell%(x*y))/x;
	if(!py) py = 30; cell -= py;
	pz = cell/(x*y);
	return point(px, py, pz);
}

__global__ void d_create_box(int **box_neigh, point len) {
	int tbox, box, diff = len.y*len.z + len.x, nbox;
	point jiter, temp, iter;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i <= len.prod()) {
		iter = xyz(i + diff, len);
		nbox = 0, box = (iter - point(0, 1, 1)).cell(len);
		jiter = iter - point(3, 3, 3);
		for(int j = 1; j <= 343; j++) {
			tbox = (cmod(jiter, len) - point(0, 1, 1)).cell(len);
			if(tbox != box) box_neigh[++nbox][box] = tbox;
			jiter.next(iter + point(3, 3, 3), point(1, 1, 1), iter - point(3, 3, 3));
		}
	}
}

void create_box() {
	int thr = 256, blk = (len.prod() + thr - 1)/thr;
	nbox = 342;
	d_create_box<<<blk, thr>>>(box_neigh, len);
}

__global__ void d_neighbour_list_md(int **neighbour, int *n_neighbour, point *pos_colloid, int no_of_colloid, double sig_colloid, point len) {
	double neigh_cutoff = 3.0*sig_colloid;
	point temp;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i < no_of_colloid) {
		for(int j = i + 1; j <= no_of_colloid; j++) {
			temp = img(pos_colloid[i] - pos_colloid[j], len);  
			if((temp*temp).sum() < power(neigh_cutoff,2)) 
				neighbour[++n_neighbour[i]][i] = j;
		}
	}
}
void neighbour_list_md() {
	int thr = 256, blk = (no_of_colloid + thr - 1)/thr;
	hipMemset(n_neighbour, 0, sizeof(int)*(no_of_colloid + 2));  
	d_neighbour_list_md<<<blk, thr>>>(neighbour, n_neighbour, pos_colloid, no_of_colloid, sig_colloid, len);
}

__global__ void d_boxpart(int **box_part, int *fluid_no, int no_of_fluid, point *pos_fl, point len)  {
	int box_no;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_fluid) {
		box_no = 1 + pos_fl[i].cell(len);
		box_part[box_no][atomicAdd(&(fluid_no[box_no]), 1) + 1] = i;
	}
}
__global__ void neighfl_sync(int **neigh_fl, int *no_neigh, int no_of_colloid) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_colloid)
		thrust::sort(thrust::seq, neigh_fl[i] + 1, neigh_fl[i] + no_neigh[i] + 1);
}

__global__ void d_neighbour_list_mpcd(int **box_part, int *fluid_no, int **box_neigh, int **neigh_fl, int *no_neigh,
	point *pos_colloid, point *pos_fl, int no_of_fluid, int no_of_colloid, int nbox, point len) {
	int mm, cbox;
	int j = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(j <= no_of_colloid) {
		no_neigh[j] = 0;
		cbox = 1 + pos_colloid[j].cell(len);
		for(int k = 1; k <= nbox; k++) {
			mm = box_neigh[k][cbox];
			for(int i = 1; i <= fluid_no[mm]; i++) {
				neigh_fl[j][atomicAdd(&no_neigh[j], 1) + 1] = box_part[mm][i];
			}
		}
	}
}


void neighbour_list_mpcd() {
	int thr = 512, blk = (no_of_fluid + thr - 1)/thr;
	hipMemset(fluid_no, 0, sizeof(int)*(len.prod() + 2));
	d_boxpart<<<blk, thr>>>(box_part, fluid_no, no_of_fluid, pos_fl, len);
	blk = (len.prod() + thr - 1)/thr;
	blk = (no_of_colloid + thr - 1)/thr;
	d_neighbour_list_mpcd<<<blk, thr>>>(box_part, fluid_no, box_neigh, neigh_fl, no_neigh, pos_colloid, pos_fl, no_of_fluid, no_of_colloid, nbox, len);
	neighfl_sync<<<blk, thr>>>(neigh_fl, no_neigh, no_of_colloid);
}
