#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <hip/hip_runtime_api.h>

__global__ void d_tumble(point *ra, point *pos_colloid, point len, int no_of_colloid, int *iv, int *seed, int *idum, int *iy){
	for(int i = 1; i <= no_of_colloid; i++) {
		ra[i] = img(pos_colloid[i] - ra[i].random(iv, seed, idum, iy)*len, len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
	}
}

void tumble() {
	d_tumble<<<1, 1>>>(ra, pos_colloid, len, no_of_colloid, iv, seed, idum, iy);
}

__global__ void d_nbrc(point *ra, point *vel_colloid, point *pos_fl, point *pos_colloid, point len, 
		int *no_neigh, int **nbr, int **neigh_fl, int *cnt, int no_of_colloid, double v0, double sigma) {
	point vector;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
//	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if(i <= no_of_colloid) {
		vel_colloid[i] += ra[i]*v0;
		cnt[i] = 0;
		for(int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);
			if((vector*vector).sum() <= power(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[atomicAdd(&cnt[i], 1) + 1][i] = neigh_fl[i][j];
		}
	}
}
__global__ void d_velc(point *ra, point *vel_fl, int **nbr, int *cnt, int no_of_colloid, double mass_colloid, double mass_fl, double v0) {
	point del; double temp;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if(i <= no_of_colloid) {
		del = ra[i]*v0;
		if(j <= cnt[i]) {
			temp = mass_colloid/(mass_fl*cnt[i]);
			vel_fl[nbr[j][i]] = vel_fl[nbr[j][i]] - del*temp;
		}
	}
}
void run() {
	dim3 thrs(32, 32), blks;
	blks = dim3((no_of_colloid + thrs.x -1)/thrs.x, (10000 + thrs.y)/thrs.y); // 10000 is no_neigh max value
	int thr = 256, blk = (thr + no_of_colloid -1)/thr;
	d_nbrc<<<blk, thr>>>(ra, vel_colloid, pos_fl, pos_colloid, len, no_neigh, nbr, neigh_fl, cnt, no_of_colloid, v0, sigma);
	d_velc<<<blks, thrs>>>(ra, vel_fl, nbr, cnt, no_of_colloid, mass_colloid, mass_fl, v0);
}
void updown_velocity() {
	point up_vel = point(0, 0, 0), vector, vel;
	for (int i = 1; i <= no_of_colloid; i++){
		cnt[i] = 0, up_cnt[i] = 0, vel = point(0, 0, 0);
		for (int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				nbr[++cnt[i]][i] = neigh_fl[i][j];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_nbr[++up_cnt[i]][i] = neigh_fl[i][j];
		}
		for (int j = 1; j <= cnt[i]; j++)
			vel += vel_fl[nbr[j][i]];

		for(int j = 1; j <= up_cnt[i]; j++)
			up_vel += vel_fl[up_nbr[j][i]];

		up_vel = (up_cnt[i] > 0)? up_vel/up_cnt[i] - vel_colloid[i]: up_vel;
		vel    = (up_cnt[i] > 0)? vel/cnt[i] - vel_colloid[i]: vel;
	}
}
