#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <hip/hip_runtime_api.h>

__global__ void d_tumble(point *ra, point *pos_colloid, point len, int no_of_colloid, int *iv, int *seed, int *idum, int *iy){
	for(int i = 1; i <= no_of_colloid; i++) {
		ra[i] = img(pos_colloid[i] - ra[i].random(iv, seed, idum, iy)*len, len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
	}
}

void tumble() {
	d_tumble<<<1, 1>>>(ra, pos_colloid, len, no_of_colloid, iv, seed, idum, iy);
}

__global__ void d_nbrc(point *ra, point *vel_colloid, point *pos_fl, point *pos_colloid, point len, 
		int *no_neigh, int **nbr, int **neigh_fl, int *cnt, int no_of_colloid, double v0, double sigma) {
	point vector;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
//	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
//	for(int i = 1; i <= no_of_colloid; i++) {
	if(i <= no_of_colloid) {
		vel_colloid[i] += ra[i]*v0;
		cnt[i] = 0;
		for(int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);
			if((vector*vector).sum() <= power(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[atomicAdd(&cnt[i], 1) + 1][i] = neigh_fl[i][j];
		}
	}
}
void d_velc(point *ra, point *vel_fl, int **nbr, int *cnt, int no_of_colloid, double mass_colloid, double mass_fl, double v0) {
	point del; double temp;
	for(int i = 1; i <= no_of_colloid; i++) {
		del = ra[i]*v0;
		for(int j = 1; j <= cnt[i]; j++) {
			temp = mass_colloid/(mass_fl*cnt[i]);
			//atomic Sub Double supported in tesla P100;
			vel_fl[nbr[j][i]] = vel_fl[nbr[j][i]] - del*temp;
		}
	}
}

void run() {
	dim3 thr(32), blk((no_of_colloid + thr.x -1)/thr.x);
	hipDeviceSynchronize();
	d_nbrc<<<blk, thr>>>(ra, vel_colloid, pos_fl, pos_colloid, len, no_neigh, nbr, neigh_fl, cnt, no_of_colloid, v0, sigma);
	hipDeviceSynchronize();
	d_velc(ra, vel_fl, nbr, cnt, no_of_colloid, mass_colloid, mass_fl, v0);
}
void updown_velocity() {
	point up_vel = point(0, 0, 0), vector, vel;
	for (int i = 1; i <= no_of_colloid; i++){
		cnt[i] = 0, up_cnt[i] = 0, vel = point(0, 0, 0);
		for (int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				nbr[++cnt[i]][i] = neigh_fl[i][j];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_nbr[++up_cnt[i]][i] = neigh_fl[i][j];
		}
		for (int j = 1; j <= cnt[i]; j++)
			vel += vel_fl[nbr[j][i]];

		for(int j = 1; j <= up_cnt[i]; j++)
			up_vel += vel_fl[up_nbr[j][i]];

		up_vel = (up_cnt[i] > 0)? up_vel/up_cnt[i] - vel_colloid[i]: up_vel;
		vel    = (up_cnt[i] > 0)? vel/cnt[i] - vel_colloid[i]: vel;
	}
}
