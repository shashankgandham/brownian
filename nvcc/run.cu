#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <hip/hip_runtime_api.h>

__global__ void d_tumble(point *ra, point *pos_colloid, point len, int no_of_colloid, int *iv, int *seed, int *idum, int *iy){
	for(int i = 1; i <= no_of_colloid; i++) {
		ra[i] = img(pos_colloid[i] - ra[i].random(iv, seed, idum, iy)*len, len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
	}
}

void tumble() {
	d_tumble<<<1, 1>>>(ra, pos_colloid, len, no_of_colloid, iv, seed, idum, iy);
}

void d_nbrc(point *ra, point *vel_colloid, point *pos_fl, point *pos_colloid, point len, 
		int *no_neigh, int **nbr, int **neigh_fl, int *cnt, int no_of_colloid, double v0, double sigma) {
	point vector;
//	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	for(int i = 1; i <= no_of_colloid; i++) {
		vel_colloid[i] += ra[i]*v0;
		cnt[i] = 0;
		for(int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);
			if((vector*vector).sum() <= power(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[++cnt[i]][i] = neigh_fl[i][j];
		}
	}
}
void d_velc(point *ra, point *vel_fl, int **nbr, int *cnt, int no_of_colloid, double mass_colloid, double mass_fl, double v0) {
	point del; double temp;
	for(int i = 1; i <= no_of_colloid; i++) {
		del = ra[i]*v0;
		for(int j = 1; j <= cnt[i]; j++) {
			temp = mass_colloid/(mass_fl*cnt[i]);
			vel_fl[nbr[j][i]] = vel_fl[nbr[j][i]] - del*temp;
		}
	}
}

void run() {
//	int thr = 256, blk = (no_of_colloid + thr -1)/thr;
	hipDeviceSynchronize();
	d_nbrc(ra, vel_colloid, pos_fl, pos_colloid, len, no_neigh, nbr, neigh_fl, cnt, no_of_colloid, v0, sigma);
	d_velc(ra, vel_fl, nbr, cnt, no_of_colloid, mass_colloid, mass_fl, v0);
}
void updown_velocity() {
	point up_vel = point(0, 0, 0), vector, vel;
	for (int i = 1; i <= no_of_colloid; i++){
		cnt[i] = 0, up_cnt[i] = 0, vel = point(0, 0, 0);
		for (int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				nbr[++cnt[i]][i] = neigh_fl[i][j];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_nbr[++up_cnt[i]][i] = neigh_fl[i][j];
		}
		for (int j = 1; j <= cnt[i]; j++)
			vel += vel_fl[nbr[j][i]];

		for(int j = 1; j <= up_cnt[i]; j++)
			up_vel += vel_fl[up_nbr[j][i]];

		up_vel = (up_cnt[i] > 0)? up_vel/up_cnt[i] - vel_colloid[i]: up_vel;
		vel    = (up_cnt[i] > 0)? vel/cnt[i] - vel_colloid[i]: vel;
	}
}
