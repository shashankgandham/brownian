#include "hip/hip_runtime.h"
#include "parameters.cuh"

__global__ void d_tumble(point *ra, point *pos_colloid, point len, int no_of_colloid){
	for (int i = 1; i <= no_of_colloid; i++) {
		ra[i] = img(pos_colloid[i] - ra[i].random(point(0, 0, 0), len), len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
    }
}

void tumble() {
	d_tumble<<<1, 1>>>(ra, pos_colloid, len, no_of_colloid);
}

__global__ void d_run(point *ra, point *vel_colloid, point *vel_fl, point *pos_fl, point *pos_colloid, point len, 
				int *no_neigh, int **nbr, int **neigh_fl, int *cnt, int *up_cnt, int no_of_colloid, double mass_fl, 
				double v0, double mass_colloid, double sigma) {
	point vector, del;
	double temp;
	for(int i = 1; i <= no_of_colloid; i++) {
		vel_colloid[i] += ra[i]*v0, del = ra[i]*v0;
		cnt[i] = up_cnt[i] = 0;
		for(int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[j][i]] - pos_colloid[i], len);
			if((vector*vector).sum() <= power(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[++cnt[i]][i] = neigh_fl[j][i];
 		}
		for(int j = 1; j <= cnt[i]; j++) {
			temp = mass_colloid/(mass_fl*cnt[i]);
			vel_fl[nbr[j][i]] = vel_fl[nbr[j][i]] - del*temp;
		}
	}
}

void run() {
	d_run<<<1, 1>>>(ra, vel_colloid, vel_fl, pos_fl, pos_colloid, len, no_neigh, nbr, neigh_fl, 
		cnt, up_cnt, no_of_colloid, mass_fl, v0, mass_colloid, sigma);
}
void updown_velocity() {
	point up_vel = point(0, 0, 0), vector, vel;
	for (int i = 1; i <= no_of_colloid; i++){
		cnt[i] = 0, up_cnt[i] = 0, vel = point(0, 0, 0);
		for (int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[j][i]] - pos_colloid[i], len);
			
			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				nbr[++cnt[i]][i] = neigh_fl[j][i];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_nbr[++up_cnt[i]][i] = neigh_fl[j][i];
		}
		for (int j = 1; j <= cnt[i]; j++)
			vel += vel_fl[nbr[j][i]];

		for(int j = 1; j <= up_cnt[i]; j++)
			up_vel += vel_fl[up_nbr[j][i]];

		up_vel = (up_cnt[i] > 0)? up_vel/up_cnt[i] - vel_colloid[i]: up_vel;
		vel    = (up_cnt[i] > 0)? vel/cnt[i] - vel_colloid[i]: vel;
	}
}