#include "parameters.cuh"
int main() {
	point mom = point(0, 0, 0);
	initialize();
	initialize_colloid();
	initialize_fluid();
	create_box();
	neighbour_list_mpcd();
	neighbour_list_md();
	compute_force_md();
	tumble();
	printf(" After Tumble\n");
	for(nn = 1; nn <= niter; nn++) {
		printf("%12d\n", nn);
		rotation_mpcd();
		run();
		for(int l = 1; l <= n; l++) {
			update_pos_md();
			neighbour_list_md();
			update_pos_mpcd();
			neighbour_list_mpcd();
			if(!(l%10) && nn > 10000) updown_velocity();
			fluid_colloid_collision();
			update_activity_direction();
			compute_force_md();
			update_velocity_colloid();
		}
		hipDeviceSynchronize();
		for(int i = 1; i <= no_of_colloid; i++) {
			pos_colloid[i].print();
			vel_colloid[i].print();
			ang_vel_colloid[i].print();
		}
	}
	return 0;
}
