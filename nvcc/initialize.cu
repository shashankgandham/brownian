#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <cstring>
point *pos_colloid, *pos_fl, *vel_colloid, *vel_fl, *ang_vel_colloid, *f, *ra, *old_force, len = point(30, 30, 30), *cell_vel, **rot, *dump_vel_fl, **u;
int n = 10, niter = 21000, file = 0, nbin = 300, maxpart = 100, no_of_colloid = 10, nbox, **nbr, **up_nbr, *cnt, *up_cnt, *fluid_no, *iv, *seed, *iy;
int no_of_fluid = len.prod()*10, *no_neigh, **neigh_fl, **neighbour, *n_neighbour, **box_neigh, **box_part, **cell_part, nn, ran_c = 0, *idum;

double kbt = 1, kbt1 = 1, ndt = 0.1, dv = 0.1, mass_fl = 1.0, mass_colloid = 654.1, sig_colloid = 5.0, eps = 1.0, v0 = 0.04;
double dt = ndt/(double)n, sigma = 0.80*sig_colloid, I_colloid = 0.1*mass_colloid*sigma*sigma, *potential_colloid, *rana, *ranb;

void initialize() {
	point **ppointers[]  = {&pos_fl, &vel_fl, &f, &pos_colloid, &vel_colloid, &ang_vel_colloid, &old_force, &ra};
	int   **ipointers[]  = {&fluid_no, &n_neighbour, &no_neigh, &cnt, &up_cnt};
	int isize[]          = {(int)len.prod(), no_of_colloid };
	int psize[]          = {no_of_fluid, no_of_colloid};
	hipMallocManaged(&box_part,  (len.prod() + 2)*sizeof(int *));
	hipMallocManaged(&cell_part, (len.prod() + 2)*sizeof(int *));
	hipMallocManaged(&rot, (len.prod() + 2)*sizeof(point *));
	hipMallocManaged(&cell_vel, (len.prod() + 2)*sizeof(point));
	hipMallocManaged(&dump_vel_fl, (no_of_fluid + 2)*sizeof(point));
	hipMallocManaged(&nbr, 7005*sizeof(int *));
	hipMallocManaged(&up_nbr, 7005*sizeof(int *));
	hipMallocManaged(&box_neigh, sizeof(int *)*512);
	hipMallocManaged(&neighbour, sizeof(int *)*256);
	hipMallocManaged(&neigh_fl,  sizeof(int *)*(no_of_colloid + 2));
	hipMallocManaged(&u,  sizeof(point *)*(no_of_colloid + 2));
	hipMallocManaged(&iv, sizeof(int)*64);
	hipMallocManaged(&seed, sizeof(int));
	hipMallocManaged(&idum, sizeof(int));
	hipMallocManaged(&iy, sizeof(int));
	hipMallocManaged(&potential_colloid, sizeof(double));
	hipMallocManaged(&rana, sizeof(double)*(len.prod() + 2));
	hipMallocManaged(&ranb, sizeof(double)*(len.prod() + 2));
	*seed = 77777, *idum = 123456789, *iy = 0;
	for(int i = 0; i < 64; i++) iv[i] = 0;
	for(int i = 0; i < 8; i++) {
		if(i < 5)  hipMallocManaged(ipointers[i], (isize[i>0] + 2)*sizeof(int));
		hipMallocManaged(ppointers[i], (psize[i>1] + 2)*sizeof(point));
	}
	for(int i = 0; i <= len.prod(); i++) {
		if(i <= 500)       hipMallocManaged(&box_neigh[i], sizeof(int)*(len.prod()    + 2));
		if(i <= 200)       hipMallocManaged(&neighbour[i], sizeof(int)*(no_of_colloid + 2));
		if(i <= 7000)      hipMallocManaged(&nbr[i],       sizeof(int)*(no_of_colloid + 2));
		if(i <= 7000)      hipMallocManaged(&up_nbr[i],    sizeof(int)*(no_of_colloid + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&neigh_fl[i],  sizeof(int)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&u[i],  sizeof(point)*(10000 + 2));
							hipMallocManaged(&box_part[i],  sizeof(int)*(maxpart    + 2));
						   hipMallocManaged(&cell_part[i], sizeof(int)*(maxpart    + 2));
						   hipMallocManaged(&rot[i],		sizeof(point)*4);
	}
}
void initialize_colloid() {
	int counter = 0, check, nofp = 0;
	double space_limit = 1.3*sig_colloid, ang_vscale_colloid = sqrt(12.0*kbt1/I_colloid), vscale_colloid = sqrt(12.0*kbt1/mass_colloid);
	point avr_vel = point(0, 0, 0), t, temp, iter = point(4, 4, 4), lim = len - point(1, 1, 1);

	for(int i = 0; i <= lim.prod(); i += 5, iter.next(lim, point(5, 5, 5), point(4, 4, 4)), nofp++) {
		if(nofp < no_of_colloid) pos_colloid[++nofp] = iter;
		else break;
	}
	while(counter < no_of_colloid) {
		t = t.random(iv, seed, idum, iy)*len;
		check = 1;
		for(int j = 1; j <= counter; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < space_limit)? 0: check;
		}
		if(check)
			pos_colloid[++counter] = t;
	}
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = (vel_colloid[j].random(iv, seed, idum, iy) - point(0.5, 0.5, 0.5))*vscale_colloid;
		avr_vel += vel_colloid[j];
	}
	avr_vel = avr_vel/no_of_colloid;
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = vel_colloid[j] - avr_vel;
		ang_vel_colloid[j] = (t.random(iv, seed, idum, iy) - point(0.5, 0.5, 0.5))*ang_vscale_colloid;
	}
}

void initialize_fluid() {
	int counter = 0, check;
	double vscale_fluid = sqrt(12.0*kbt/mass_fl);
	point avr_vel = point(0, 0, 0), t, temp;
	while(counter < no_of_fluid) {
		t = t.random(iv, seed, idum, iy)*len;
		check = 1;
		for(int j = 1; j <= no_of_colloid; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < sigma*0.5)? 0: check;
		}
		if(check) 
			pos_fl[++counter] = t;
	}
	for(int j = 1; j <= no_of_fluid; j++) {
		vel_fl[j] = (vel_fl[j].random(iv, seed, idum, iy) - point(0.5, 0.5, 0.5))*vscale_fluid;
		avr_vel += vel_fl[j];
	}
	avr_vel = avr_vel/no_of_fluid;
	for(int j = 1; j <= no_of_fluid; j++) {
		vel_fl[j] = vel_fl[j] - avr_vel;
	}
}