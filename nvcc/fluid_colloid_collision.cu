#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline CUDA_CALLABLE_MEMBER point crossmul(point a, point b) {
	return point(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

inline CUDA_CALLABLE_MEMBER point stochastic_reflection(point rf, point rs, double mass_fl, double kbt, point len, int *iv, int *seed, int *idum, int *iy) {
	double m_beta = mass_fl/kbt, random_e = power(1 - ran(iv, seed, idum, iy), 2), val, v[4], x[4], z = 2;
	point un, ut, t, n;
	n = rs/sqrt((rs*rs).sum());
	val = sqrt(-log(random_e)/m_beta);

	un = n*val;
	t = img((t.random(iv, seed, idum, iy)*len - rf), len);
	ut = crossmul(un, t);
	ut = ut/sqrt((ut*ut).sum());
	while(z > 1) {
		x[1] = 2.0 * ran(iv, seed, idum, iy) - 1, x[2] = 2.0 * ran(iv, seed, idum, iy) - 1;
		z = x[1]*x[1] + x[2]*x[2];
	}
	z = sqrt((-2.0*log(z))/z);
	v[1] = x[1]*z*sqrt(kbt/mass_fl); v[2] = x[2]*z*sqrt(kbt/mass_fl);
	return ut*v[1] + un;
}

void d_fluid_colloid_collision(int *no_neigh, point *pos_colloid, point *pos_fl, point *vel_colloid, 
		point *ang_vel_colloid, point *dump_vel_fl, double mass_colloid, double I_colloid,
		double mass_fl, double dt, point *vel_fl, point len, double sigma, int no_of_colloid,
		double kbt, int **neigh_fl, int *iv, int *seed, int *idum, int *iy) {
	point rr, rs, u, omega, vc;
	for(int j = 1; j <= no_of_colloid; j++) {
		vc = omega = point(0, 0, 0);
		for(int i = 1; i <= no_neigh[j]; i++) {
			int l = neigh_fl[i][j];
			rr = img(pos_colloid[j] - pos_fl[l], len);
			if((rr*rr).sum() <= pow(sigma, 2)*0.25) {
				pos_fl[l] = mod(pos_fl[l] - vel_fl[l]*dt* 0.5, len);
				rs = img(pos_fl[l] - pos_colloid[j], len);
				u  = stochastic_reflection(pos_fl[l], rs, mass_fl, kbt, len, iv, seed, idum, iy);
				vel_fl[l] = u + vel_colloid[j] + crossmul(ang_vel_colloid[j], rs);
				vc += (dump_vel_fl[l] - vel_fl[l]);
				u = (dump_vel_fl[l] - vel_fl[l]);
				omega += crossmul(rs, (dump_vel_fl[l] - vel_fl[l]));
				pos_fl[l] = mod(pos_fl[l] + vel_fl[l]*dt*0.5, len);
			}
		}
		vel_colloid[j] 	   += vc*mass_fl/mass_colloid;
		ang_vel_colloid[j] += omega*mass_fl/I_colloid;
	}
}
__global__ void d_dump(point *dump_vel_fl, point *vel_fl, int no_of_fluid) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_fluid) dump_vel_fl[i] = vel_fl[i];
}
void fluid_colloid_collision() {
	int thr = 256, blk = (no_of_fluid + thr - 1)/thr;
	d_dump<<<blk, thr>>> (dump_vel_fl, vel_fl, no_of_fluid);
	hipDeviceSynchronize();
	d_fluid_colloid_collision (no_neigh, pos_colloid, pos_fl, vel_colloid, ang_vel_colloid, dump_vel_fl, 
			mass_colloid, I_colloid, mass_fl, dt, vel_fl, len, sigma, no_of_colloid, kbt, neigh_fl, iv, seed, idum, iy);
}
