#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline __device__ point crossmul(point a, point b) {
	return point(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

inline __device__ point stochastic_reflection(point rf, point rs, double mass_fl, double kbt, point len, int *iv, int *seed, int *idum, int *iy) {
	double m_beta = mass_fl/kbt, random_e = power(1 - ran(iv, seed, idum, iy), 2), val, v[4], x[4], z = 2;
	point un, ut, t, n;
	n = rs/sqrt((rs*rs).sum());
	val = sqrt(-log(random_e)/m_beta);

	un = n*val;
	t = img((t.random(iv, seed, idum, iy)*len - rf), len);
	ut = crossmul(un, t);
	ut = ut/sqrt((ut*ut).sum());
	while(z > 1) {
		x[1] = 2.0 * ran(iv, seed, idum, iy) - 1, x[2] = 2.0 * ran(iv, seed, idum, iy) - 1;
		z = x[1]*x[1] + x[2]*x[2];
	}
	z = sqrt((-2.0*log(z))/z);
	v[1] = x[1]*z*sqrt(kbt/mass_fl); v[2] = x[2]*z*sqrt(kbt/mass_fl);
	return ut*v[1] + un;
}
__global__ void d_fluid_colloid_collision(int *no_neigh, point *pos_colloid, point *pos_fl, point *vel_colloid, 
		point *ang_vel_colloid, point *dump_vel_fl, double mass_colloid, double I_colloid,
		double mass_fl, double dt, point *vel_fl, point len, double sigma, int no_of_colloid,
		double kbt, int **neigh_fl, int *iv, int *seed, int *idum, int *iy) {
	point rr, rs, u, omega, vc;
	for(int j = 1; j <= no_of_colloid; j++) {
		vc = omega = point(0, 0, 0);
		for(int i = 1; i <= no_neigh[j]; i++) {
			int l = neigh_fl[i][j];
			rr = img(pos_colloid[j] - pos_fl[l], len);
			if((rr*rr).sum() <= pow(sigma, 2)*0.25) {
				pos_fl[l] = mod(pos_fl[l] - vel_fl[l]*dt* 0.5, len);
				rs = img(pos_fl[l] - pos_colloid[j], len);
				u  = stochastic_reflection(pos_fl[l], rs, mass_fl, kbt, len, iv, seed, idum, iy);
				vel_fl[l] = u + vel_colloid[j] + crossmul(ang_vel_colloid[j], rs);
				vc += (dump_vel_fl[l] - vel_fl[l]);
				u = (dump_vel_fl[l] - vel_fl[l]);
				omega += crossmul(rs, (dump_vel_fl[l] - vel_fl[l]));
				pos_fl[l] = mod(pos_fl[l] + vel_fl[l]*dt*0.5, len);
			}
		}
		vel_colloid[j] 	   += vc*mass_fl/mass_colloid;
		ang_vel_colloid[j] += omega*mass_fl/I_colloid;
	}
}

void fluid_colloid_collision() {
	point *dump_vel_fl;
	hipMallocManaged(&dump_vel_fl, sizeof(point)*(no_of_fluid + 2));
	hipMemcpy(dump_vel_fl, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
	d_fluid_colloid_collision<<<1, 1>>> (no_neigh, pos_colloid, pos_fl, vel_colloid, ang_vel_colloid, dump_vel_fl, 
			mass_colloid, I_colloid, mass_fl, dt, vel_fl, len, sigma, no_of_colloid, kbt, neigh_fl, iv, seed, idum, iy);
}
