#include "hip/hip_runtime.h"
#include "parameters.cuh"

__device__ double d_asin(double x) { return asin(x); }

__global__ void d_rotation_mpcd(point *vel_fl, point *pos_fl, int *fluid_no, int **cell_part, int no_of_fluid, 
						point len, double kbt, double mass_fl, int *iv, int *seed, int *idum, int *iy) {
	int k, cell_no;
	double r[4], ir[4], theta, phi, rho, var, scale_fac_mpcd, ct, st, ict;
	point *cell_vel, del_v, rr, rot[4], temp;
	cell_vel = (point *)malloc((len.prod() + 2)*sizeof(point));
	memset(fluid_no, 0, (len.prod() + 2)*sizeof(int));
	rr = rr.random(iv, seed, idum, iy) - point(0.5, 0.5, 0.5);
	for(int i = 1; i <= no_of_fluid; i++) {
		temp = mod(pos_fl[i] + rr, len);
		cell_no = 1 + temp.cell(len);
		cell_part[++fluid_no[cell_no]][cell_no] = i;
	}
	memset(cell_vel, 0, sizeof cell_vel);
	for(int i = 1; i <= len.prod(); i++) {
		if (fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++)
				cell_vel[i] += vel_fl[cell_part[j][i]]/fluid_no[i];
			rho = 2*ran(iv, seed, idum, iy) - 1, phi = 4.0*d_asin(1.0)*ran(iv, seed, idum, iy), theta = 2*d_asin(1)*(130/180.0);
			r[1] = cos(phi)*sqrt(1 - rho*rho), r[2] = sin(phi)*sqrt(1 - rho*rho), r[3] = rho;
			ct = cos(theta), st = sin(theta), ict = 1 - cos(theta);
			ir[1] = ict*r[1], ir[2] = ict*r[2], ir[3] = ict*r[3];
			rot[1] = point(ir[1]*r[1] + ct, 	 ir[1]*r[2] - st*r[3], ir[1]*r[3] + st*r[2]);
			rot[2] = point(ir[2]*r[1] + st*r[3], ir[2]*r[2] + ct, 	   ir[2]*r[3] - st*r[1]);
			rot[3] = point(ir[3]*r[1] - st*r[2], ir[3]*r[2] + st*r[1], ir[3]*r[3] + ct);
			for(int j = 1;j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + point((rot[1]*del_v).sum(), (rot[2]*del_v).sum(), (rot[3]*del_v).sum());
			}
		}
	}
	for(int i = 1; i <= len.prod(); i++) {
		var = 0.0;
		if(fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				var += (del_v*del_v).sum();
			}
			scale_fac_mpcd = sqrt(3.0 * (fluid_no[i] - 1) * kbt/(mass_fl * var));
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + del_v*scale_fac_mpcd;
			}
		}
	}
}

void rotation_mpcd() {
	d_rotation_mpcd<<<1, 1>>> (vel_fl, pos_fl, fluid_no, cell_part, no_of_fluid, len, kbt, mass_fl, iv, seed, idum, iy);
}
