#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

__device__ double sinx(double n) {
    double denominator, sinx, x1;
    n = n * (M_PI/ 180.0);
    x1 = n, sinx = n;
    for(int i = 1; i <= 15; i++) {
        denominator = 2 * i * (2 * i + 1);
        x1 = -x1 * n * n / denominator;
        sinx = sinx + x1;
    }
	return sinx;
}
__device__ double cosx(double n) {
    double x1, denominator, cosx;
    n = n * (M_PI / 180.0), x1 = 1;
    cosx = x1;
    for(int i = 1; i<= 15; i++) {
		denominator = 2 * i * (2 * i - 1);
        x1 = -x1 * n * n / denominator;
        cosx = cosx + x1;
    }
    return cosx;
}

__global__ void cellpart_sync(int **cell_part, int *fluid_no, point len) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= len.prod())
		thrust::sort(thrust::seq, cell_part[i] + 1, cell_part[i] + fluid_no[i] + 1);
}
__global__ void d_cellpart(int **cell_part, int *fluid_no, int no_of_fluid, point *pos_fl, point rr, point len) {
	int cell_no, i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	point temp;
	if(i <= no_of_fluid) {
		cell_no = mod(pos_fl[i] + rr, len).cell(len) + 1;
		cell_part[cell_no][atomicAdd(&(fluid_no[cell_no]), 1) + 1] = i;
	}
}
__global__ void  d_cellvel(point *cell_vel, point *vel_fl, int **cell_part, int *fluid_no, point len) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= len.prod() && fluid_no[i] > 1) {
		for(int	j = 1; j <= fluid_no[i]; j++)
				cell_vel[i] += vel_fl[cell_part[i][j]]/fluid_no[i];
	}

}
__global__ void  d_velfl(point *cell_vel, point *vel_fl, int **cell_part, int *fluid_no, point *rot[4], point len) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	point del_v;
	int k;
	if(i <= len.prod() && fluid_no[i] > 1) {
		for(int	j = 1; j <= fluid_no[i]; j++) {
			k = cell_part[i][j];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + point((rot[i][1]*del_v).sum(), (rot[i][3]*del_v).sum(), (rot[i][3]*del_v).sum());
		}
	}

}

__global__ void d_rotation_mpcd(point *vel_fl, point *pos_fl, point *cell_vel, point **rot, int *fluid_no, int **cell_part, int no_of_fluid, 
		point len, double kbt, double mass_fl, int *iv, int *seed, int *idum, int *iy) {
	double r[4], ir[4], theta, phi, rho, ct, st, ict;
	point del_v, temp;
	for(int i = 1; i <= len.prod(); i++) {
		if (fluid_no[i] > 1) {
			rho = 2*ran(iv, seed, idum, iy) - 1, phi = 2.0*M_PI*ran(iv, seed, idum, iy), theta = M_PI*(130/180.0);
			r[1] = cosx(phi)*sqrt(1 - rho*rho), r[2] = sinx(phi)*sqrt(1 - rho*rho), r[3] = rho;
			ct = cosx(theta), st = sinx(theta), ict = 1 - cosx(theta);
			ir[1] = ict*r[1], ir[2] = ict*r[2], ir[3] = ict*r[3];
			rot[i][1] = point(ir[1]*r[1] + ct, 	 ir[1]*r[2] - st*r[3], ir[1]*r[3] + st*r[2]);
			rot[i][2] = point(ir[2]*r[1] + st*r[3], ir[2]*r[2] + ct, 	   ir[2]*r[3] - st*r[1]);
			rot[i][3] = point(ir[3]*r[1] - st*r[2], ir[3]*r[2] + st*r[1], ir[3]*r[3] + ct);
		}
	}
}
__global__ void d_rotate(int *fluid_no, int**cell_part, point *vel_fl, point *cell_vel, point len, double mass_fl, double kbt) {
	point del_v;
	double var, scale_fac_mpcd;
	int k, i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= len.prod()) {
		var = 0.0;
		if(fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[i][j];
				del_v = vel_fl[k] - cell_vel[i];
				var += (del_v*del_v).sum();
			}
			scale_fac_mpcd = sqrt(3.0 * (fluid_no[i] - 1) * kbt/(mass_fl * var));
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[i][j];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + del_v*scale_fac_mpcd;
			}
		}
	}
}
__global__ void getRandom(point *rr, int *iv, int *seed, int *idum, int *iy) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(!i) *rr = (*rr).random(iv, seed, idum, iy) - point(0.5, 0.5, 0.5);
}

void rotation_mpcd() {
	point rr;
	int thr = 256, blk = (no_of_fluid + thr -1)/thr;
	getRandom<<<2, 2>>>(&rr, iv, seed, idum, iy);
	hipMemset(cell_vel, 0, sizeof cell_vel);
	hipMemset(fluid_no, 0, (len.prod() + 2)*sizeof(int));
	d_cellpart<<<blk, thr>>>(cell_part, fluid_no, no_of_fluid, pos_fl, rr, len);
	blk = (len.prod() + thr - 1)/thr;
	cellpart_sync<<<blk, thr>>> (cell_part, fluid_no, len);
	d_cellvel<<<blk, thr>>>(cell_vel, vel_fl, cell_part, fluid_no, len);
	d_rotation_mpcd<<<1, 1>>> (vel_fl, pos_fl, cell_vel, rot, fluid_no, cell_part, no_of_fluid, len, kbt, mass_fl, iv, seed, idum, iy);
	d_velfl<<<blk, thr>>>(cell_vel, vel_fl, cell_part, fluid_no, rot, len);
	d_rotate<<<blk, thr>>> (fluid_no, cell_part, vel_fl, cell_vel, len, mass_fl, kbt);
}
