#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

__global__ void cellpart_sync(int **cell_part, int *fluid_no, point len) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= len.prod())
		thrust::sort(thrust::seq, cell_part[i] + 1, cell_part[i] + fluid_no[i] + 1);
}
__global__ void d_cellpart(int **cell_part, int *fluid_no, int no_of_fluid, point *pos_fl, point rr, point len) {
	int cell_no, i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	point temp;
	if(i <= no_of_fluid) {
		cell_no = mod(pos_fl[i] + rr, len).cell(len) + 1;
		cell_part[cell_no][atomicAdd(&(fluid_no[cell_no]), 1) + 1] = i;
	}
}
__global__ void  d_cellvel(point *cell_vel, point *vel_fl, int **cell_part, int *fluid_no, point len) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= len.prod() && fluid_no[i] > 1) {
		for(int	j = 1; j <= fluid_no[i]; j++)
				cell_vel[i] += vel_fl[cell_part[i][j]]/fluid_no[i];
	}

}
__global__ void  d_velfl(point *cell_vel, point *vel_fl, int **cell_part, int *fluid_no, point *rot[4], point len) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	point del_v;
	int k;
	if(i <= len.prod() && fluid_no[i] > 1) {
		for(int	j = 1; j <= fluid_no[i]; j++) {
			k = cell_part[i][j];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + point((rot[i][1]*del_v).sum(), (rot[i][2]*del_v).sum(), (rot[i][3]*del_v).sum());
		}
	}

}


void d_rotation_mpcd(point *vel_fl, point *pos_fl, point *cell_vel, point **rot, int *fluid_no, int **cell_part, int no_of_fluid, 
		point len, double kbt, double mass_fl, double *rana, double *ranb) {
	double r[4], ir[4], theta, phi, rho, ct, st, ict;
	point del_v, temp;
//	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	for(int i = 1; i <= len.prod(); i++) {
//	if(i <= len.prod()) {
		if (fluid_no[i] > 1) {
			rho = 2*rana[i] - 1, phi = 2.0*M_PI*ranb[i], theta = M_PI*(130/180.0);
			r[1] = cos(phi)*sqrt(1 - rho*rho), r[2] = sin(phi)*sqrt(1 - rho*rho), r[3] = rho;
			ct = cos(theta), st = sin(theta), ict = 1 - cos(theta);
			ir[1] = ict*r[1], ir[2] = ict*r[2], ir[3] = ict*r[3];
			rot[i][1] = point(ir[1]*r[1] + ct, 	 ir[1]*r[2] - st*r[3], ir[1]*r[3] + st*r[2]);
			rot[i][2] = point(ir[2]*r[1] + st*r[3], ir[2]*r[2] + ct, 	   ir[2]*r[3] - st*r[1]);
			rot[i][3] = point(ir[3]*r[1] - st*r[2], ir[3]*r[2] + st*r[1], ir[3]*r[3] + ct);
		}
	}
}
__global__ void d_rotate(int *fluid_no, int**cell_part, point *vel_fl, point *cell_vel, point len, double mass_fl, double kbt) {
	point del_v;
	double var, scale_fac_mpcd;
	int k, i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= len.prod()) {
		var = 0.0;
		if(fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[i][j];
				del_v = vel_fl[k] - cell_vel[i];
				var += (del_v*del_v).sum();
			}
			scale_fac_mpcd = sqrt(3.0 * (fluid_no[i] - 1) * kbt/(mass_fl * var));
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[i][j];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + del_v*scale_fac_mpcd;
			}
		}
	}
}

void rotation_mpcd() {
	point rr;
	int thr = 256, blk = (no_of_fluid + thr -1)/thr;
	
	hipDeviceSynchronize();
	rr = rr.random(iv, seed, idum, iy) - point(0.5, 0.5, 0.5);
	hipMemset(cell_vel, 0, (len.prod() + 2)*sizeof(point));
	hipMemset(fluid_no, 0, (len.prod() + 2)*sizeof(int));
	d_cellpart<<<blk, thr>>>(cell_part, fluid_no, no_of_fluid, pos_fl, rr, len);
	
	blk = (len.prod() + thr - 1)/thr;
	cellpart_sync<<<blk, thr>>> (cell_part, fluid_no, len);
	d_cellvel<<<blk, thr>>>(cell_vel, vel_fl, cell_part, fluid_no, len);

	hipDeviceSynchronize();
	for(int  i = 1; i <= len.prod(); i++) {
		if(fluid_no[i] > 1) {
			rana[i] = ran(iv, seed, idum, iy);
			ranb[i] = ran(iv, seed, idum, iy);
		}
	}
 
	d_rotation_mpcd(vel_fl, pos_fl, cell_vel, rot, fluid_no, cell_part, no_of_fluid, len, kbt, mass_fl, rana, ranb);
	d_velfl<<<blk, thr>>>(cell_vel, vel_fl, cell_part, fluid_no, rot, len);
	d_rotate<<<blk, thr>>> (fluid_no, cell_part, vel_fl, cell_vel, len, mass_fl, kbt);
}
