#include "hip/hip_runtime.h"
#include "parameters.cuh"

double dtb2 = dt/(mass_colloid*2);

__global__ void d_compute_force_md(point *f, int *n_neighbour, int **neighbour, point *pos_colloid, double sig_colloid, double eps, double *potential_colloid, point len, int no_of_colloid) {
	double mag_f, r_cutoff, fc, ufc, sig_colloid12, sig_colloid6, r;
	r_cutoff = power(2, 1.0/6.0)*sig_colloid, r, fc = 4.0*eps*(12.0*(power(sig_colloid,12)/power(r_cutoff,13)) - 6.0*(power(sig_colloid, 6)/power(r_cutoff, 7)));
	ufc = 4.0*eps*(power(sig_colloid/r_cutoff, 12) - power(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;
	sig_colloid12 = power(sig_colloid, 12), sig_colloid6 = power(sig_colloid, 6);
	point temp, ff;
	double t1, t2;
	for(int i = 1; i <= no_of_colloid; i++) {
		for(int j = 1; j <= n_neighbour[i]; j++) {
			temp = img(pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
			r = sqrt((temp*temp).sum());
			if(r < r_cutoff) {
				*potential_colloid += 4*eps*(power(sig_colloid/r, 12) - power(sig_colloid/r, 6)) - ufc + fc*r;
				t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
				mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
				ff = (temp*mag_f)/r;
				f[i] += ff, f[neighbour[j][i]] -= ff;
			}   
		}
	}
}
__global__ void d_update_activity_direction(point *ang_vel_colloid, point *ra, double dt, int no_of_colloid) {
	point m[4], b, sb, cb;
	for(int i = 1; i <= no_of_colloid; i++) {
		b  = ang_vel_colloid[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));
		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		m[2] =  point((sb.x*sb.y)*cb.z + cb.x*sb.z, (-sb.x*sb.y)*sb.z + cb.x*cb.z, -sb.x*cb.y);
		m[3] =  point((-cb.x*sb.y)*cb.z + sb.x*sb.z, (cb.x*sb.y)*sb.z + sb.x*cb.z, cb.x*cb.y);
		ra[i] = point((m[1]*ra[i]).sum(), (m[2]*ra[i]).sum(), (m[3]*ra[i]).sum());
	}
}
__global__ void d_update_pos_md(point *pos_colloid, point *vel_colloid, point *f, double dt, double mass_colloid, point len, int no_of_colloid) {
	double dt2= dt*dt, ddt = 0.5*dt2/mass_colloid;
	for(int i = 1; i <= no_of_colloid; i++)
		pos_colloid[i] = mod(pos_colloid[i] + vel_colloid[i]*dt + f[i]*ddt, len);
}
__global__ void d_update_pos_mpcd(point *pos_fl, point *vel_fl, double dt, point len, int no_of_fluid) {
	for(int i = 1; i <= no_of_fluid; i++)
	pos_fl[i] = mod(pos_fl[i] + vel_fl[i]*dt, len);
}
__global__ void d_update_vel_colloid(point *vel_colloid, point *old_force, point *f, double dtb2, int no_of_colloid){
	for(int i = 1; i <= no_of_colloid; i++) 
		vel_colloid[i] += old_force[i] + f[i]*dtb2; 
}

void compute_force_md() { 
	double *d_potential_colloid;
	hipMalloc(&d_potential_colloid, sizeof(double));
	hipMemcpy(d_potential_colloid, &potential_colloid, sizeof(double), hipMemcpyHostToDevice);	
	d_compute_force_md<<<1, 1>>>(f, n_neighbour, neighbour, pos_colloid, sig_colloid, eps, d_potential_colloid, len, no_of_colloid);
	hipMemcpy(&potential_colloid, d_potential_colloid, sizeof(double), hipMemcpyHostToDevice);	

}

void update_activity_direction() {
	d_update_activity_direction<<<1, 1>>>(ang_vel_colloid, ra, dt, no_of_colloid);
}
void update_pos_md() {
	d_update_pos_md<<<1, 1>>>(pos_colloid, vel_colloid, f, dt, mass_colloid, len, no_of_colloid);
}

void update_pos_mpcd() {
	d_update_pos_mpcd<<<1, 1>>>(pos_fl, vel_fl, dt, len, no_of_fluid);
}

void update_velocity_colloid() {
	d_update_vel_colloid<<<1, 1>>>(vel_colloid, old_force, f, dtb2, no_of_colloid);
}
