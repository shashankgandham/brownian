#include "hip/hip_runtime.h"
#include "parameters.cuh"
	
double sig_colloid12 = pow(sig_colloid, 12), sig_colloid6 = pow(sig_colloid, 6);
double r_cutoff = pow(2, 1.0/6.0)*sig_colloid, r, fc = 0; //4.0*eps*(12.0*(sig_colloid12/pow(r_cutoff,13)) - 6.0*(sig_colloid6/pow(r_cutoff, 7)));
double ufc = 4.0*eps*(pow(sig_colloid/r_cutoff, 12) - pow(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;

void d_compute_force_md(point *f, int *n_neighbour, int **neighbour, point *pos_colloid, double sig_colloid, double sig_colloid12, double sig_colloid6, double r_cutoff, double fc, double ufc, double eps, double *potential_colloid, point len, int no_of_colloid) {
	point temp, ff;
	double t1, t2, mag_f = 0, r;
//	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
//	if(i <= no_of_colloid) {
	for(int i = 1; i <= no_of_colloid; i++) {
		for(int j = 1; j <= n_neighbour[i]; j++) {
			temp = img(pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
			r = sqrt((temp*temp).sum());
			if(r < r_cutoff) {
				*potential_colloid += 4*eps*(power(sig_colloid/r, 12) - power(sig_colloid/r, 6)) - ufc + fc*r;
				t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
				mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
				ff = (temp*mag_f)/r;
				f[i] += ff, f[neighbour[j][i]] -= ff;
			}   
		}
	}
}

void compute_force_md() {
//	int thr = 256, blk = (no_of_colloid + thr - 1)/thr; 
	//hipDeviceSynchronize();
	for(int i = 1; i <= no_of_colloid; i++) f[i] = point(0, 0, 0);
	d_compute_force_md(f, n_neighbour, neighbour, pos_colloid, sig_colloid, sig_colloid12, sig_colloid6, r_cutoff, fc, ufc, eps, potential_colloid, len, no_of_colloid);
}
__global__ void d_update_activity_direction(point *ang_vel_colloid, point *ra, double dt, int no_of_colloid) {
	point m[4], b, sb, cb;
	for(int i = 1; i <= no_of_colloid; i++) {
		b  = ang_vel_colloid[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));
		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		m[2] =  point((sb.x*sb.y)*cb.z + cb.x*sb.z, (-sb.x*sb.y)*sb.z + cb.x*cb.z, -sb.x*cb.y);
		m[3] =  point((-cb.x*sb.y)*cb.z + sb.x*sb.z, (cb.x*sb.y)*sb.z + sb.x*cb.z, cb.x*cb.y);
		ra[i] = point((m[1]*ra[i]).sum(), (m[2]*ra[i]).sum(), (m[3]*ra[i]).sum());
	}
}

void update_activity_direction() {
	d_update_activity_direction<<<1, 1>>>(ang_vel_colloid, ra, dt, no_of_colloid);
}

__global__ void d_update_pos_md(point *pos_colloid, point *vel_colloid, point *f, point *old_force, double dt, double ddt, double mass_colloid, point len, int no_of_colloid) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_colloid) {
		old_force[i] = f[i];
		pos_colloid[i] += vel_colloid[i]*dt + f[i]*ddt;
		pos_colloid[i] = mod(pos_colloid[i], len);
	}
}
void update_pos_md() {
	int thr = 256, blk = (no_of_colloid + thr - 1)/thr;
	double dt2= dt*dt, ddt = 0.5*dt2/mass_colloid;
	d_update_pos_md<<<blk, thr>>>(pos_colloid, vel_colloid, f, old_force, dt, ddt, mass_colloid, len, no_of_colloid);
}

__global__ void d_update_pos_mpcd(point *pos_fl, point *vel_fl, double dt, point len, int no_of_fluid) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;	
	if(i <= no_of_fluid) {
		pos_fl[i] = mod(pos_fl[i] + vel_fl[i]*dt, len);
	}
}
void update_pos_mpcd() {
	int thr = 256, blk = (no_of_fluid + thr - 1)/thr;
	d_update_pos_mpcd<<<blk, thr>>>(pos_fl, vel_fl, dt, len, no_of_fluid);
}

__global__ void d_update_vel_colloid(point *vel_colloid, point *old_force, point *f, double dtb2, int no_of_colloid) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;	
	if(i <= no_of_colloid) vel_colloid[i] += (old_force[i] + f[i])*dtb2; 
}
void update_velocity_colloid() {
	double dtb2 = dt/(mass_colloid*2);
	int thr = 256, blk = (no_of_colloid + thr - 1)/thr;
	d_update_vel_colloid<<<blk, thr>>>(vel_colloid, old_force, f, dtb2, no_of_colloid);
}
