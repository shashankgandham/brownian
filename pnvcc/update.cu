#include "hip/hip_runtime.h"
#include "parameters.cuh"

double mag_f, r_cutoff = pow(2, 1.0/6.0)*sig_colloid, r;
double fc = 4.0*eps*(12.0*(pow(sig_colloid,12)/pow(r_cutoff,13)) - 6.0*(pow(sig_colloid, 6)/pow(r_cutoff, 7)));
double ufc = 4.0*eps*(pow(sig_colloid/r_cutoff, 12) - pow(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;
double sig_colloid12 = pow(sig_colloid, 12), sig_colloid6 = pow(sig_colloid, 6);

double power(double x, int r) {
    double ans = 1;
    for(int i = 1; i <=r; i++)
        ans *= x;
    return ans;
}
void compute_force_md() {
	point temp, ff;
	potential_colloid = 0;
    memset(f, 0, sizeof(int)*(no_of_colloid + 2));
    for(int i = 0; i <= 10; i++) f[i] = 0;
    double t1, t2, t3, rp;
	for(int i = 1; i <= no_of_colloid; i++) {
		for(int j = 1; j <= n_neighbour[i]; j++) {
			temp = img(pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
			r = sqrt((temp*temp).sum());
			if(r < r_cutoff) {
				potential_colloid += 4*eps*(pow(sig_colloid/r, 12) - pow(sig_colloid/r, 6)) - ufc + fc*r;
				t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
                mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
                ff = (temp*mag_f)/r;
				f[i] += ff, f[neighbour[j][i]] -= ff;
            }
		}   
	}
}

__global__ void d_update_activity(int no_of_colloid, point *d_ang_vel, point *d_ra, double dt){
	point m[4], b, sb, cb;
	for (int i = 1; i <= no_of_colloid; i++){
		b  = d_ang_vel[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));

		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		m[2] =  point(sb.x*sb.y*cb.z + cb.x*sb.z, -sb.x*sb.y*sb.z + cb.x*cb.z, -sb.x*cb.y);
		m[3] =  point(-cb.x*sb.y*cb.z + sb.x*sb.z, cb.x*sb.y*sb.z + sb.x*cb.z, cb.x*cb.y);
		d_ra[i] = point((m[1]*d_ra[i]).sum(), (m[2]*d_ra[i]).sum(), (m[3]*d_ra[i]).sum());
	}
}


void update_activity_direction() {
	point m[4], b, sb, cb;
	for (int i = 1; i <= no_of_colloid; i++) {
		b  = ang_vel_colloid[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));

		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		m[2] =  point(sb.x*sb.y*cb.z + cb.x*sb.z, -sb.x*sb.y*sb.z + cb.x*cb.z, -sb.x*cb.y);
		m[3] =  point(-cb.x*sb.y*cb.z + sb.x*sb.z, cb.x*sb.y*sb.z + sb.x*cb.z, cb.x*cb.y);
		ra[i] = point((m[1]*ra[i]).sum(), (m[2]*ra[i]).sum(), (m[3]*ra[i]).sum());
		ra[i].print();
   		
	}
	/*
    point *d_ang_vel, *d_ra;

    hipMalloc(&d_ang_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_ra, (no_of_colloid + 2)*sizeof(point));
    
    hipMemcpy(d_ang_vel, ang_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_ra, ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	
  	d_update_activity<<<1,1>>>(no_of_colloid, d_ang_vel, d_ra, dt);

  	hipMemcpy(ra, d_ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_ra); hipFree(d_ang_vel);

  	for (int i = 1; i <= no_of_colloid; i++)
  		ra[i].print();
  	*/
}

__global__ void cuda_update_md(point *cuda_pos, point *cuda_vel, point *cuda_f, double dt, double mass_colloid){
	double dt2 = dt * dt, ddt = 0.5*dt2/mass_colloid;
	cuda_pos[blockIdx.x] += cuda_vel[blockIdx.x]*dt + cuda_f[blockIdx.x]*ddt; 

}

//PARALLELIZED: MOD REMAINING
void update_pos_md() {
	//pos_colloid[no_of_colloid].print();
	/*double dt2 = dt*dt, ddt = 0.5*dt2/mass_colloid;
	for(int i = 1; i <= no_of_colloid; i++) {
		pos_colloid[i] +=  vel_colloid[i]*dt + f[i]*ddt;
		pos_colloid[i]  =  mod(pos_colloid[i], len);
    	pos_colloid[i].print();
    }*/

    point *cuda_pos, *cuda_vel, *cuda_f;
    hipMalloc(&cuda_pos, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&cuda_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&cuda_f, (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(cuda_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(cuda_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(cuda_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	cuda_update_md<<<no_of_colloid + 1,1>>>(cuda_pos,cuda_vel,cuda_f,dt,mass_colloid);
  	hipMemcpy(pos_colloid, cuda_pos, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(cuda_pos); hipFree(cuda_vel); hipFree(cuda_f);	

}

__global__ void d_update_mpcd(int n, point *d_pos, point *d_vel, double dt, point len){
	int i = blockIdx.x*blockDim.x + threadIdx.x;
  	if (i <= n) 
  		d_pos[i] = d_pos[i] + d_vel[i]*dt;
	
}

void update_pos_mpcd() {
	for (int i = 1; i <= no_of_fluid; i++) {
		pos_fl[i] = mod(pos_fl[i] + vel_fl[i]*dt, len);
		//pos_fl[i].print();
	}
	/*point *d_pos, *d_vel;
    hipMalloc(&d_pos, (no_of_fluid + 2)*sizeof(point));
    hipMalloc(&d_vel, (no_of_fluid + 2)*sizeof(point));
    hipMemcpy(d_pos, pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_vel, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	
  	d_update_mpcd<<<(no_of_fluid + 255)/256,256>>>(no_of_fluid, d_pos,d_vel,dt,len);
  	
  	hipMemcpy(pos_fl, d_pos, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_pos); hipFree(d_vel); 
  	for (int i = 1; i <= no_of_fluid; i++){
  		pos_fl[i] = mod(pos_fl[i], len);
  		pos_fl[i].print();
  	}*/

}

__global__ void d_update_vel_colloid(point *d_vel, point *d_old_force, point *d_f, double dtb2){
	d_vel[blockIdx.x] += d_old_force[blockIdx.x] + d_f[blockIdx.x]*dtb2; 
}

//PARALLELIZED
void update_velocity_colloid() {
    double dtb2 = dt/(mass_colloid*2);
	/*for (int i = 1; i <= no_of_colloid; i++) {
		vel_colloid[i] += (old_force[i] + f[i])*dtb2;
		vel_colloid[i].print();		
    }*/

    point *d_vel, *d_old_force, *d_f;
    hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_old_force, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_f, (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(d_old_force, old_force, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	d_update_vel_colloid<<<no_of_colloid + 1,1>>>(d_vel, d_old_force, d_f, dtb2);

  	hipMemcpy(vel_colloid, d_vel, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_old_force); hipFree(d_vel); hipFree(d_f);

}
