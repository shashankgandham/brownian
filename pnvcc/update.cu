#include "hip/hip_runtime.h"
#include "parameters.cuh"
//update pos md is parallelized
//update_vel_colloid is parallelized
double mag_f, r_cutoff = pow(2, 1.0/6.0)*sig_colloid, r;
double fc = 4.0*eps*(12.0*(pow(sig_colloid,12)/pow(r_cutoff,13)) - 6.0*(pow(sig_colloid, 6)/pow(r_cutoff, 7)));
double ufc = 4.0*eps*(pow(sig_colloid/r_cutoff, 12) - pow(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;
double sig_colloid12 = pow(sig_colloid, 12), sig_colloid6 = pow(sig_colloid, 6);

double power(double x, int r) {
    double ans = 1;
    for(int i = 1; i <=r; i++)
        ans *= x;
    return ans;
}
void compute_force_md() {
	point temp, ff;
	potential_colloid = 0;
    for(int i = 0; i <= no_of_colloid; i++) f[i] = 0;
    double t1, t2, t3, rp;
	for(int i = 1; i <= no_of_colloid; i++) {
		for(int j = 1; j <= n_neighbour[i]; j++) {
			temp = img(pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
			r = sqrt((temp*temp).sum());
			if(r < r_cutoff) {
				potential_colloid += 4*eps*(pow(sig_colloid/r, 12) - pow(sig_colloid/r, 6)) - ufc + fc*r;
				t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
                mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
                ff = (temp*mag_f)/r;
				f[i] += ff, f[neighbour[j][i]] -= ff;
            }
		}   
	}
}

__global__ void d_update_activity(int no_of_colloid, point *d_ang_vel, point *d_ra, double dt){
	point m[4], b, sb, cb, temp;
	for (int i = 1; i <= no_of_colloid; i++){
		b  = d_ang_vel[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));
		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		temp.x = sb.x*sb.y, temp.y = -sb.x*sb.y;
		m[2] =  point(temp.x*cb.z + cb.x*sb.z, temp.y*sb.z + cb.x*cb.z, -sb.x*cb.y);
		temp.x = -cb.x*sb.y, temp.y = cb.x*sb.y;
		m[3] =  point(temp.x*cb.z + sb.x*sb.z, temp.y*sb.z + sb.x*cb.z, cb.x*cb.y);
		temp = d_ra[i];
		d_ra[i] = point((m[1]*temp).sum(), (m[2]*temp).sum(), (m[3]*temp).sum());
		
	}
}

//FLOATING POINT ERROR
void update_activity_direction() {
	point m[4], b, sb, cb, temp;
	for (int i = 1; i <= no_of_colloid; i++) {
		b  = ang_vel_colloid[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));
		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		temp.x = sb.x*sb.y, temp.y = -sb.x*sb.y;
		m[2] =  point(temp.x*cb.z + cb.x*sb.z, temp.y*sb.z + cb.x*cb.z, -sb.x*cb.y);
		temp.x = cb.x*sb.y, temp.y = cb.x*sb.y;
		m[3] =  point(sb.x*sb.z - temp.x*cb.z, temp.y*sb.z + sb.x*cb.z, cb.x*cb.y);
		temp = ra[i];
		ra[i] = point((m[1]*temp).sum(), (m[2]*temp).sum(), (m[3]*temp).sum());
		
	}

	/*	
    point *d_ang_vel, *d_ra;

    hipMalloc(&d_ang_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_ra, (no_of_colloid + 2)*sizeof(point));
    
    hipMemcpy(d_ang_vel, ang_vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_ra, ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	
  	d_update_activity<<<1,1>>>(no_of_colloid, d_ang_vel, d_ra, dt);

  	hipMemcpy(ra, d_ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_ra); hipFree(d_ang_vel);

  	for (int i = 1; i <= no_of_colloid; i++)
  		ra[i].print();
  	*/
}

__global__ void d_update_md(point *cuda_pos, point *cuda_vel, point *cuda_f, double dt, double mass_colloid, point len){
	double dt2 = dt * dt, ddt = 0.5*dt2/mass_colloid;
	d_mod(&cuda_pos[blockIdx.x], cuda_pos[blockIdx.x] += cuda_vel[blockIdx.x]*dt + cuda_f[blockIdx.x]*ddt, len);

}

//PARALLELIZED
void update_pos_md() {
	point *cuda_pos, *cuda_vel, *cuda_f, *mod;
    hipMalloc(&mod, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&cuda_pos, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&cuda_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&cuda_f, (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(cuda_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(cuda_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(cuda_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	d_update_md<<<no_of_colloid + 1,1>>>(cuda_pos,cuda_vel,cuda_f,dt,mass_colloid, len);
  	hipMemcpy(pos_colloid, cuda_pos, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(cuda_pos); hipFree(cuda_vel); hipFree(cuda_f);	
}


__global__ void d_update_mpcd(point *d_pos, point *d_vel, double dt, point len){
	d_mod(&d_pos[blockIdx.x], d_pos[blockIdx.x] += d_vel[blockIdx.x]*dt, len);	
}

//ERROR
void update_pos_mpcd() {
	/*for (int i = 1; i <= no_of_fluid; i++) {
		pos_fl[i] = mod(pos_fl[i] += vel_fl[i] * dt, len);
		pos_fl[i].print();
	}
	*/
	point *d_pos, *d_vel;
	
	
    hipMalloc(&d_pos, (no_of_fluid + 2)*sizeof(point));
    hipMalloc(&d_vel, (no_of_fluid + 2)*sizeof(point));
    hipMemcpy(d_pos, pos_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_vel, vel_fl, (no_of_fluid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	
  	d_update_mpcd<<<(no_of_fluid +1)/4096,4096>>>(d_pos,d_vel,dt,len);
  	hipDeviceSynchronize();

  	hipMemcpy(pos_fl, d_pos, (no_of_fluid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_pos); hipFree(d_vel); 
  	
}

__global__ void d_update_vel_colloid(point *d_vel, point *d_old_force, point *d_f, double dtb2){
	d_vel[blockIdx.x] += d_old_force[blockIdx.x] + d_f[blockIdx.x]*dtb2; 
}

//PARALLELIZED
void update_velocity_colloid() {
    double dtb2 = dt/(mass_colloid*2);
	
    point *d_vel, *d_old_force, *d_f;
    hipMalloc(&d_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_old_force, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_f, (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(d_old_force, old_force, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	d_update_vel_colloid<<<no_of_colloid + 1,1>>>(d_vel, d_old_force, d_f, dtb2);

  	hipMemcpy(vel_colloid, d_vel, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_old_force); hipFree(d_vel); hipFree(d_f);
 }