#include "hip/hip_runtime.h"
#include "parameters.cuh"

double mag_f, r_cutoff = pow(2, 1.0/6.0)*sig_colloid, r;
double fc = 4.0*eps*(12.0*(pow(sig_colloid,12)/pow(r_cutoff,13)) - 6.0*(pow(sig_colloid, 6)/pow(r_cutoff, 7)));
double ufc = 4.0*eps*(pow(sig_colloid/r_cutoff, 12) - pow(sig_colloid/r_cutoff, 6)) + fc*r_cutoff;
double sig_colloid12 = pow(sig_colloid, 12), sig_colloid6 = pow(sig_colloid, 6);

double power(double x, int r) {
    double ans = 1;
    for(int i = 1; i <=r; i++)
        ans *= x;
    return ans;
}
void compute_force_md() {
	point temp, ff;
	potential_colloid = 0;
    memset(f, 0, sizeof(int)*(no_of_colloid + 2));
    for(int i = 0; i <= 10; i++) f[i] = 0;
    double t1, t2, t3, rp;
	for(int i = 1; i <= no_of_colloid; i++) {
		for(int j = 1; j <= n_neighbour[i]; j++) {
			temp = img(pos_colloid[i] - pos_colloid[neighbour[j][i]], len);
			r = sqrt((temp*temp).sum());
			if(r < r_cutoff) {
				potential_colloid += 4*eps*(pow(sig_colloid/r, 12) - pow(sig_colloid/r, 6)) - ufc + fc*r;
				t1 = sig_colloid12/power(r,13), t2 = sig_colloid6/power(r, 7);
                mag_f = 4.0*eps*(12.0*t1 - 6.0*t2) - fc;
                ff = (temp*mag_f)/r;
				f[i] += ff, f[neighbour[j][i]] -= ff;
            }
		}   
	}
}

void update_activity_direction() {
	point m[4], b, sb, cb;
	for (int i = 1; i <= no_of_colloid; i++) {
		b  = ang_vel_colloid[i]*dt;
		sb = point(sin(b.x), sin(b.y), sin(b.z)), cb = point(cos(b.x), cos(b.y), cos(b.z));

		m[1] =  point(cb.y*cb.z, -cb.y*sb.z, sb.y);
		m[2] =  point(sb.x*sb.y*cb.z + cb.x*sb.z, -sb.x*sb.y*sb.z + cb.x*cb.z, -sb.x*cb.y);
		m[3] =  point(-cb.x*sb.y*cb.z + sb.x*sb.z, cb.x*sb.y*sb.z + sb.x*cb.z, cb.x*cb.y);
		ra[i] = point((m[1]*ra[i]).sum(), (m[2]*ra[i]).sum(), (m[3]*ra[i]).sum());
	
    }
}

__global__ void cuda_update(point *cuda_pos, point *cuda_vel, point *cuda_f, double dt, double mass_colloid){
	double dt2 = dt * dt, ddt = 0.5*dt2/mass_colloid;
	cuda_pos[blockIdx.x] += cuda_vel[blockIdx.x]*dt + cuda_f[blockIdx.x]*ddt; 

}

void update_pos_md() {
	//pos_colloid[no_of_colloid].print();
	/*double dt2 = dt*dt, ddt = 0.5*dt2/mass_colloid;
	for(int i = 1; i <= no_of_colloid; i++) {
		pos_colloid[i] +=  vel_colloid[i]*dt + f[i]*ddt;
		pos_colloid[i]  =  mod(pos_colloid[i], len);
    	pos_colloid[i].print();
    }*/

    point *cuda_pos, *cuda_vel, *cuda_f;
    hipMalloc(&cuda_pos, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&cuda_vel, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&cuda_f, (no_of_colloid + 2)*sizeof(point)); 
  	hipMemcpy(cuda_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(cuda_vel, vel_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(cuda_f, f, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);  	
  	
  	cuda_update<<<no_of_colloid + 1,1>>>(cuda_pos,cuda_vel,cuda_f,dt,mass_colloid);
  	hipMemcpy(pos_colloid, cuda_pos, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(cuda_pos); hipFree(cuda_vel); hipFree(cuda_f);	
  	for(int i = 1; i <= no_of_colloid; i++) {
		pos_colloid[i].print();
    }

}

void update_pos_mpcd() {
	for (int i = 1; i <= no_of_fluid; i++) {
		pos_fl[i] = mod(pos_fl[i] + vel_fl[i]*dt, len);
	}
}

void update_velocity_colloid() {
    double dtb2 = dt/(mass_colloid*2);
	for (int i = 1; i <= no_of_colloid; i++) {
		vel_colloid[i] += (old_force[i] + f[i])*dtb2;
//	    if(nn == 369) vel_colloid[i].print();
  //      if(nn == 369) old_force[i].print();
    //    if(nn == 369) f[i].print();
    }
}
