#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include "hip/hip_runtime.h"

void initialize() {
	point **ppointers[]  = {&pos_fl, &vel_fl, &f, &pos_colloid, &vel_colloid, &ang_vel_colloid, &old_force, &ra};
	int   **ipointers[]  = {&fluid_no, &n_neighbour, &no_neigh, &cnt, &up_cnt};
	int isize[]          = {len.prod(), no_of_colloid };
	int psize[]          = {no_of_fluid, no_of_colloid};

	//calloc((int**)&box_part, )
	box_part 	= (int **)calloc((maxpart + 2), sizeof(int *));
	cell_part 	= (int **)calloc((maxpart + 2), sizeof(int *));
	nbr 		= (int **)calloc(7005,sizeof(int *));
	up_nbr 		= (int **)calloc(7005,sizeof(int *));
    iv = (int *)calloc((ntab + 2), sizeof(int));

	for(int i = 0; i < 8; i++) {
		if(i < 5) *ipointers[i] = (int   *)calloc(isize[i>0] + 2, sizeof(int)  );
				  *ppointers[i] = (point *)calloc(psize[i>1] + 2, sizeof(point));

	}
	for(int i = 0; i <= 10000; i++) {
		if(i <= 500)      box_neigh[i] = (int *)calloc(sizeof(int),(len.prod()    + 2));
		if(i <= maxpart)  box_part[i]  = (int *)calloc(sizeof(int),(len.prod()    + 2));
		if(i <= maxpart)  cell_part[i] = (int *)calloc(sizeof(int),(len.prod()    + 2));
		if(i <= 200)      neighbour[i] = (int *)calloc(sizeof(int),(no_of_colloid + 2));
		if(i <= 7000)     nbr[i]       = (int *)calloc(sizeof(int),(no_of_colloid + 2));
		if(i <= 7000)     up_nbr[i]    = (int *)calloc(sizeof(int),(no_of_colloid + 2));
						  neigh_fl[i]  = (int *)calloc(sizeof(int),(no_of_colloid + 2));
	}
}

void initialize_colloid() {
	int counter = 0, check, nofp = 0, x = len.x, y = len.y, z = len.z;
	double space_limit = 1.3*sig_colloid, ang_vscale_colloid = sqrt(12.0*kbt1/I_colloid), vscale_colloid = sqrt(12.0*kbt1/mass_colloid);
	point avr_vel = point(0, 0, 0), t, temp, iter = point(4, 4, 4), lim = len - point(1, 1, 1);

	for(int i = 0; i <= lim.prod(); i += 5, iter.next(lim, point(5, 5, 5), point(4, 4, 4)), nofp++) {
		if(nofp < no_of_colloid) pos_colloid[nofp] = iter;
		else break;
	}
    int tempx = 0;
	while(counter < no_of_colloid) {
		t = t.random(point(0, 0, 0), len);
		check = 1;
		for(int j = 1; j <= counter; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < space_limit)? 0: check;
        }
		if(check)
			pos_colloid[++counter] = t;
	}
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = vel_colloid[j].random(point(0.5, 0.5, 0.5))*vscale_colloid;
		avr_vel += vel_colloid[j];
	}
	avr_vel = avr_vel/no_of_colloid;
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = vel_colloid[j] - avr_vel;
		ang_vel_colloid[j] = (t.random(point(0.5, 0.5, 0.5)))*ang_vscale_colloid;
	}
}

void initialize_fluid() {
	int counter = 0, check;
	double vscale_fluid = sqrt(12.0*kbt/mass_fl);
	point avr_vel = point(0, 0, 0), t, temp;
	while(counter < no_of_fluid) {
		t = t.random(point(0, 0, 0), len);
		check = 1;
		for(int j = 1; j <= no_of_colloid; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < sigma*0.5)? 0: check;
		}
		if(check)
			pos_fl[++counter] = t;
	}
	for(int j = 1; j <= no_of_fluid; j++) {
		vel_fl[j] = vel_fl[j].random(point(0.5, 0.5, 0.5))*vscale_fluid;
		avr_vel += vel_fl[j];
	}
	avr_vel = avr_vel/no_of_fluid;
	for(int j = 1; j <= no_of_fluid; j++)
		vel_fl[j] = vel_fl[j] - avr_vel;
}