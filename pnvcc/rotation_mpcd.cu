#include "hip/hip_runtime.h"
#include "parameters.cuh"


void rotation_mpcd() {
	int k, cell_no;
	double r[4], ir[4], theta, phi, rho, var, scale_fac_mpcd, ct, st, ict;
	point cell_vel[(int)len.prod() + 1], del_v, rr, rot[4], temp;

	memset(fluid_no, 0, (len.prod() + 2)*sizeof(int));
	rr.random(point(0.5, 0.5, 0.5));
	for(int i = 1; i <= no_of_fluid; i++) {
		//cell_no = 1 + mod(pos_fl[i] + rr, len).cell(len);
		d_mod(&temp, pos_fl[i] + rr, len);
		cell_no = temp.cell(len) + 1;
		cell_part[++fluid_no[cell_no]][cell_no] = i;
	}
	memset(cell_vel, 0, sizeof cell_vel);
	for(int i = 1; i <= len.prod(); i++) {
		if (fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++)
				cell_vel[i] += vel_fl[cell_part[j][i]]/fluid_no[i];
			rho = 2*ran() - 1, phi = 4.0*asin(1.0)*ran(), theta = 2*asin(1)*(130/180.0);
			r[1] = cos(phi)*sqrt(1 - rho*rho), r[2] = sin(phi)*sqrt(1 - rho*rho), r[3] = rho;
			ct = cos(theta), st = sin(theta), ict = 1 - cos(theta);
			ir[1] = ict*r[1], ir[2] = ict*r[2], ir[3] = ict*r[3];
			rot[1] = point(ir[1]*r[1] + ct, 	 ir[1]*r[2] - st*r[3], ir[1]*r[3] + st*r[2]);
			rot[2] = point(ir[2]*r[1] + st*r[3], ir[2]*r[2] + ct, 	   ir[2]*r[3] - st*r[1]);
			rot[3] = point(ir[3]*r[1] - st*r[2], ir[3]*r[2] + st*r[1], ir[3]*r[3] + ct);
			for(int j = 1;j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + point((rot[1]*del_v).sum(), (rot[2]*del_v).sum(), (rot[3]*del_v).sum());
			}
		}
	}
	int m = 0;
	for(int i = 1; i <= len.prod(); i++) {
		var = 0.0;
		if(fluid_no[i] > 1) {
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				var += (del_v*del_v).sum();
			}
			scale_fac_mpcd = sqrt(3.0 * (fluid_no[i] - 1) * kbt/(mass_fl * var));
			for(int j = 1; j <= fluid_no[i]; j++) {
				k = cell_part[j][i];
				del_v = vel_fl[k] - cell_vel[i];
				vel_fl[k] = cell_vel[i] + del_v*scale_fac_mpcd;
			}
		}
	}
}
