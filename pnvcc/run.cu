#include "hip/hip_runtime.h"
#include "parameters.cuh"

__global__ void d_tumble(int no_of_colloid, point *d_ra, point *d_pos, point len){
	for (int i = 1; i <= no_of_colloid; i++) {
		//d_img(&d_pos[i],d_pos[i] - d_ra[i].random(point(0, 0, 0), len), len);
		//d_ra[i] = d_ra[i]/sqrt((d_ra[i]*d_ra[i]).sum());
		point c;
		d_random(&c, point(0, 0, 0), len, seed ,iv);
    }
}

void tumble(){
	/*for (int i = 1; i <= no_of_colloid; i++) {
		ra[i] = img(pos_colloid[i] - ra[i].random(point(0, 0, 0), len), len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
		ra[i].print();
    }*/
    
    point *d_ra, *d_pos;
    hipMalloc(&d_ra, (no_of_colloid + 2)*sizeof(point));
    hipMalloc(&d_pos, (no_of_colloid + 2)*sizeof(point));
    hipMemcpy(d_ra, ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	hipMemcpy(d_pos, pos_colloid, (no_of_colloid + 2)*sizeof(point), hipMemcpyHostToDevice);
  	
  	d_tumble<<<1,1>>>(no_of_colloid, d_ra, d_pos, len);

  	hipMemcpy(ra, d_ra, (no_of_colloid + 2)*sizeof(point), hipMemcpyDeviceToHost);
  	hipFree(d_ra); hipFree(d_pos); 
  	for (int i = 1; i <= no_of_colloid; i++)
  		ra[i].print();
}

void run() {
	point vector, del;
	double temp;
	for(int i = 1; i <= no_of_colloid; i++) {
		vel_colloid[i] += ra[i]*v0, del = ra[i]*v0;
		cnt[i] = up_cnt[i] = 0;
		for(int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[j][i]] - pos_colloid[i], len);
			if((vector*vector).sum() <= pow(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[++cnt[i]][i] = neigh_fl[j][i];
		}
		for(int j = 1; j <= cnt[i]; j++) {
			temp = mass_colloid/(mass_fl*cnt[i]);
			vel_fl[nbr[j][i]] = vel_fl[nbr[j][i]] - del*(mass_colloid/(mass_fl*cnt[i]));
		}
	}
}

void updown_velocity(){
	point up_vel = point(0, 0, 0), vector, vel;
	for (int i = 1; i <= no_of_colloid; i++){
		cnt[i] = 0, up_cnt[i] = 0, vel = point(0, 0, 0);
		for (int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[j][i]] - pos_colloid[i], len);
			
			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				nbr[++cnt[i]][i] = neigh_fl[j][i];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_nbr[++up_cnt[i]][i] = neigh_fl[j][i];
		}
		for (int j = 1; j <= cnt[i]; j++)
			vel += vel_fl[nbr[j][i]];

		for(int j = 1; j <= up_cnt[i]; j++)
			up_vel += vel_fl[up_nbr[j][i]];

		up_vel = (up_cnt[i] > 0)? up_vel/up_cnt[i] - vel_colloid[i]: up_vel;
		vel    = (up_cnt[i] > 0)? vel/cnt[i] - vel_colloid[i]: vel;
	}
}
