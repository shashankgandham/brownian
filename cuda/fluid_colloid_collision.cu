#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline CUDA_CALLABLE_MEMBER point crossmul(point a, point b) {
	return point(a.y*b.z - a.z*b.y, a.z*b.x - a.x*b.z, a.x*b.y - a.y*b.x);
}

inline __device__ point stochastic_reflection(point rf, point rs, double mass_fl, double kbt, point len, hiprandState_t *state) {
	double m_beta = mass_fl/kbt, random_e = power(1 - hiprand_uniform_double(state), 2), val, v[4], x[4], z = 2;
	point un, ut, t, n;
	n = rs/sqrt((rs*rs).sum());
	val = sqrt(-log(random_e)/m_beta);

	un = n*val;
	t = img((t.rand(state)*len - rf), len);
	ut = crossmul(un, t);
	ut = ut/sqrt((ut*ut).sum());
	while(z > 1) {
		x[1] = 2.0 * hiprand_uniform_double(state) - 1, x[2] = 2.0 * hiprand_uniform_double(state) - 1;
		z = x[1]*x[1] + x[2]*x[2];
	}
	z = sqrt((-2.0*log(z))/z);
	v[1] = x[1]*z*sqrt(kbt/mass_fl); v[2] = x[2]*z*sqrt(kbt/mass_fl);
	return ut*v[1] + un;
}

__global__ void d_fluid_colloid_collision(int *no_neigh, point *pos_colloid, point *pos_fl, 
		point *vel_colloid, point *ang_vel_colloid, point *dump_vel_fl, double mass_colloid, 
		double I_colloid, double mass_fl, double dt, point *vel_fl, point len, double sigma, 
		int no_of_colloid, double kbt, int **neigh_fl, point **vc, point **om, hiprandState_t *state) {

	point rr, rs, uu;
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1, i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(j <= no_of_colloid) {
		vc[j][0] = om[j][0] = point(0, 0, 0);
		if(i <= no_neigh[j]) {
			vc[j][i] = om[j][i] = point(0, 0, 0);
			int l = neigh_fl[j][i];
			rr = img(pos_colloid[j] - pos_fl[l], len);
			if((rr*rr).sum() <= pow(sigma, 2)*0.25) {
				pos_fl[l] = mod(pos_fl[l] - vel_fl[l]*dt* 0.5, len);
				rs = img(pos_fl[l] - pos_colloid[j], len);
				uu  = stochastic_reflection(pos_fl[l], rs, mass_fl, kbt, len, &state[i]);
				vel_fl[l] = uu + vel_colloid[j] + crossmul(ang_vel_colloid[j], rs);
				pos_fl[l] = mod(pos_fl[l] + vel_fl[l]*dt*0.5, len);
				point t1 = dump_vel_fl[l] - vel_fl[l], t2;
				t2  = crossmul(rs, t1);
				vc[j][i] = t1; om[j][i] = t2;
			}
		}
	}
}
__global__ void d_dump(point *dump_vel_fl, point *vel_fl, int no_of_fluid) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_fluid) dump_vel_fl[i] = vel_fl[i];
}
__global__ void update_fcc(point **vc, point **om, point *vel_colloid, point *ang_vel_colloid, 
		int *no_neigh, int no_of_colloid, double mass_colloid, double mass_fl, double I_colloid) {

	int j = blockIdx.x*blockDim.x + threadIdx.x;
	if(j <= no_of_colloid) {
		vc[j][0] = thrust::reduce(thrust::device, vc[j], vc[j] + no_neigh[j] + 1, point(0, 0, 0), add_point());
		om[j][0] = thrust::reduce(thrust::device, om[j], om[j] + no_neigh[j] + 1, point(0, 0, 0), add_point());
		vel_colloid[j] += vc[j][0]*mass_fl/mass_colloid;
		ang_vel_colloid[j] += om[j][0]*mass_fl/I_colloid;
	}
}
void fluid_colloid_collision() {
	blk = dim3((no_of_fluid + thr.x - 1)/thr.x);
	d_dump<<<blk, thr>>> (dump_vel_fl, vel_fl, no_of_fluid);
	blk = dim3((10000 + thrs.x - 1)/thrs.x, (no_of_colloid + thrs.y - 1)/thrs.y);
	d_fluid_colloid_collision<<<blk, thrs>>>(no_neigh, pos_colloid, pos_fl, vel_colloid, 
			ang_vel_colloid, dump_vel_fl, mass_colloid, I_colloid, mass_fl, dt, vel_fl, 
			len, sigma, no_of_colloid, kbt, neigh_fl, vc, om, state);
	blk = dim3((no_of_colloid + thr.x -1)/thr.x);
	update_fcc<<<blk, thr>>>(vc, om, vel_colloid, ang_vel_colloid, no_neigh, no_of_colloid, 
			mass_colloid, mass_fl, I_colloid);
}
