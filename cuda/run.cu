#include "hip/hip_runtime.h"
#include "parameters.cuh"

__global__ void d_tumble(point *ra, point *pos_colloid, point len, int no_of_colloid, hiprandState_t *state){
	int i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	if(i <= no_of_colloid)  {
		ra[i] = img(pos_colloid[i] - ra[i].rand(&state[i])*len, len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
	}
}

void tumble() {
	blk = dim3((no_of_colloid + thr.x - 1)/thr.x);
	d_tumble<<<blk, thr>>>(ra, pos_colloid, len, no_of_colloid, state);
}

__global__ void d_nbrc(point *ra, point *vel_colloid, point *pos_fl, point *pos_colloid, point len, 
		int *no_neigh, int **nbr, int **neigh_fl, int *cnt, int no_of_colloid, double v0, double sigma) {
	point vector;
	int j = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if(i <= no_of_colloid) {
		vel_colloid[i] += ra[i]*v0, cnt[i] = 0;
		if(j <= no_neigh[i]) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);
			if((vector*vector).sum() <= power(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[atomicAdd(&cnt[i], 1) + 1][i] = neigh_fl[i][j];
		}
	}
}
__global__ void d_velc(point *ra, point *vel_fl, int **nbr, int *cnt, int no_of_colloid, double mass_colloid, double mass_fl, double v0) {
	point del; double temp;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_colloid) {
		del = ra[i]*v0, temp = mass_colloid/(mass_fl*cnt[i]);
		for(int j = 1; j <= cnt[i]; j++) 
			vel_fl[nbr[j][i]] -= del*temp;
	}
}

void run() {
	blk = dim3((10000 + thr.x -1)/thr.x, (no_of_colloid + thr.y - 1)/thr.y);
	d_nbrc<<<blk, thr>>>(ra, vel_colloid, pos_fl, pos_colloid, len, no_neigh, nbr, neigh_fl, cnt, no_of_colloid, v0, sigma);
	blk = dim3((no_of_colloid + thr.x - 1)/thr.x);
	d_velc<<<blk, thr>>>(ra, vel_fl, nbr, cnt, no_of_colloid, mass_colloid, mass_fl, v0);
}

__global__ void d_updown_velocity(int no_of_colloid, int *cnt, int *up_cnt, int *no_neigh, int **nbr, int **up_nbr, 
			int **neigh_fl, point *pos_fl, point *pos_colloid, point *vel_colloid, point *vel_fl, point len, double sigma) {
	point up_vel = point(0, 0, 0), vector, vel;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;	
	if(i <= no_of_colloid) {
		cnt[i] = 0, up_cnt[i] = 0, vel = point(0, 0, 0);
		for (int j = 1; j <= no_neigh[i]; j++) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				nbr[++cnt[i]][i] = neigh_fl[i][j];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_nbr[++up_cnt[i]][i] = neigh_fl[i][j];
		}
		for (int j = 1; j <= cnt[i]; j++)
			vel += vel_fl[nbr[j][i]];

		for(int j = 1; j <= up_cnt[i]; j++)
			up_vel += vel_fl[up_nbr[j][i]];

		up_vel = (up_cnt[i] > 0)? up_vel/up_cnt[i] - vel_colloid[i]: up_vel;
		vel    = (up_cnt[i] > 0)? vel/cnt[i] - vel_colloid[i]: vel;
	}
}

void updown_velocity() {
	blk = dim3((no_of_colloid + thr.x - 1)/thr.x);
	d_updown_velocity<<<blk, thr>>>(no_of_colloid, cnt, up_cnt, no_neigh, nbr, up_nbr, neigh_fl, pos_fl, 
			pos_colloid, vel_colloid, vel_fl, len, sigma);
}
