#include "hip/hip_runtime.h"
#include "parameters.cuh"

__global__ void d_tumble(point *ra, point *pos_colloid, point len, int no_of_colloid, hiprandState_t *state){
	int i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	if(i <= no_of_colloid)  {
		ra[i] = img(pos_colloid[i] - ra[i].rand(&state[i])*len, len);
		ra[i] = ra[i]/sqrt((ra[i]*ra[i]).sum());
	}
}

void tumble() {
	blk = dim3((no_of_colloid + thr.x - 1)/thr.x);
	d_tumble<<<blk, thr>>>(ra, pos_colloid, len, no_of_colloid, state);
}

__global__ void d_nbrc(point *ra, point *vel_colloid, point *pos_fl, point *pos_colloid, point len, 
		int *no_neigh, int **nbr, int **neigh_fl, int *cnt, int no_of_colloid, double v0, double sigma) {
	point vector;
	int j = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int i = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if(i <= no_of_colloid) {
		vel_colloid[i] += ra[i]*v0, cnt[i] = 0;
		if(j <= no_neigh[i]) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);
			if((vector*vector).sum() <= power(sigma*0.5+0.5, 2) && (vector*vel_colloid[i]).sum() <= 0)
				nbr[atomicAdd(&cnt[i], 1) + 1][i] = neigh_fl[i][j];
		}
	}
}
__global__ void d_velc(point *ra, point *vel_fl, int **nbr, int *cnt, int no_of_colloid, double mass_colloid, double mass_fl, double v0) {
	point del; double temp;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_colloid) {
		del = ra[i]*v0, temp = mass_colloid/(mass_fl*cnt[i]);
		for(int j = 1; j <= cnt[i]; j++) 
			vel_fl[nbr[j][i]] -= del*temp;
	}
}

void run() {
	blk = dim3((10000 + thr.x -1)/thr.x, (no_of_colloid + thr.y - 1)/thr.y);
	d_nbrc<<<blk, thr>>>(ra, vel_colloid, pos_fl, pos_colloid, len, no_neigh, nbr, neigh_fl, cnt, no_of_colloid, v0, sigma);
	blk = dim3((no_of_colloid + thr.x - 1)/thr.x);
	d_velc<<<blk, thr>>>(ra, vel_fl, nbr, cnt, no_of_colloid, mass_colloid, mass_fl, v0);
}

__global__ void helper_upd(int no_of_colloid, int *cnt, int *up_cnt, int *no_neigh, point **vel, point **up_vel, 
			int **neigh_fl, point *pos_fl, point *pos_colloid, point *vel_colloid, point *vel_fl, point len, double sigma) {
	
	point vector;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;	
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;	
	if(i <= no_of_colloid) {
		if(j <= no_neigh[i]) {
			vector = img(pos_fl[neigh_fl[i][j]] - pos_colloid[i], len);

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.5), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				vel[i][atomicAdd(&cnt[i], 1) + 1] = vel_fl[neigh_fl[i][j]];

			if((vector*vector).sum() <= pow((sigma*0.5 + 0.1), 2) && (vector*vel_colloid[i]).sum() <= 0.0)
				up_vel[i][atomicAdd(&up_cnt[i], 1) + 1] = vel_fl[neigh_fl[i][j]];
		}
	}
}
__global__ void calc_upd(int no_of_colloid, int *cnt, int *up_cnt, point **vel, point **up_vel, point *vel_colloid) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;	
	if(i <= no_of_colloid) {
		vel[i][0]    = thrust::reduce(thrust::device, vel[i], vel[i] + cnt[i] + 1, point(0, 0, 0), add_point());	
		vel[i][0]    = (cnt[i])? vel[i][0]/cnt[i] - vel_colloid[i]: vel[i][0];
		up_vel[i][0] = thrust::reduce(thrust::device, up_vel[i], up_vel[i] + up_cnt[i] + 1, point(0, 0, 0), add_point());	
		up_vel[i][0] = (up_cnt[i])? up_vel[i][0]/up_cnt[i] - vel_colloid[i]: up_vel[i][0];
	}
}
void updown_velocity() {
	blk = dim3((no_of_colloid + thr.x - 1)/thr.x);
	imemset<<<blk, thr>>>(cnt, no_of_colloid);
	imemset<<<blk, thr>>>(up_cnt, no_of_colloid);
	helper_upd<<<blk, thr>>>(no_of_colloid, cnt, up_cnt, no_neigh, vel, up_vel, neigh_fl, pos_fl, 
			pos_colloid, vel_colloid, vel_fl, len, sigma);
	calc_upd<<<blk, thr>>>(no_of_colloid, cnt, up_cnt, vel, up_vel, vel_colloid);
}
