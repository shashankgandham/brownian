#include "hip/hip_runtime.h"
#include "parameters.cuh"

inline __device__ __host__ point cmod(point a, point b) {
	if(a.x <=  0) a.x += b.x;  if(a.y <=  0) a.y += b.y;
	if(a.z <=  0) a.z += b.z; if(a.x > b.x) a.x -= b.x;
	if(a.y > b.y) a.y -= b.y; if(a.z > b.z) a.z -= b.z;
	return a;
}

inline __device__ point xyz(int cell, point len) {
	int x = len.x, y = len.y, px, py, pz;
	px = (cell%(x*y))%x;
	if(!px) px = 30; cell -= px;
	py = (cell%(x*y))/x;
	if(!py) py = 30; cell -= py;
	pz = cell/(x*y);
	return point(px, py, pz);
}

__global__ void d_create_box(int **box_neigh, point len) {
	int tbox, box, diff = len.y*len.z + len.x, nbox;
	point jiter, temp, iter;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i <= len.prod()) {
		iter = xyz(i + diff, len);
		nbox = 0, box = (iter - point(0, 1, 1)).cell(len);
		jiter = iter - point(3, 3, 3);
		for(int j = 1; j <= 343; j++) {
			tbox = (cmod(jiter, len) - point(0, 1, 1)).cell(len);
			if(tbox != box) box_neigh[++nbox][box] = tbox;
			jiter.next(iter + point(3, 3, 3), point(1, 1, 1), iter - point(3, 3, 3));
		}
	}
}

void create_box() {
	blk = dim3((len.prod() + thr.x - 1)/thr.x);
	nbox = 342;
	d_create_box<<<blk, thr>>>(box_neigh, len);
}

__global__ void d_neighbour_list_md(int **neighbour, int *n_neighbour, point *pos_colloid, int no_of_colloid, double sig_colloid, point len) {
	double neigh_cutoff = 3.0*sig_colloid;
	point temp;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int j = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if(i < no_of_colloid) {
		if(j <= no_of_colloid) {
			temp = img(pos_colloid[i] - pos_colloid[j], len);  
			if((temp*temp).sum() < power(neigh_cutoff,2)) 
				neighbour[atomicAdd(&n_neighbour[i], 1) + 1][i] = j;
		}
	}
}
void neighbour_list_md() {
	blk = dim3((no_of_colloid + thrs.x - 1)/thrs.x, (no_of_colloid + thrs.y - 1)/thrs.y);
	hipMemset(n_neighbour, 0, sizeof(int)*(no_of_colloid + 2));  
	d_neighbour_list_md<<<blk, thrs>>>(neighbour, n_neighbour, pos_colloid, no_of_colloid, sig_colloid, len);
}

__global__ void d_boxpart(int **box_part, int *fluid_no, int no_of_fluid, point *pos_fl, point len)  {
	int box_no;
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= no_of_fluid) {
		box_no = 1 + pos_fl[i].cell(len);
		box_part[box_no][atomicAdd(&(fluid_no[box_no]), 1) + 1] = i;
	}
}

__global__ void d_neighbour_list_mpcd(int **box_part, int *fluid_no, int **box_neigh, int **neigh_fl, int *no_neigh, int **dp, 
		point *pos_colloid, point *pos_fl, int no_of_fluid, int no_of_colloid, int nbox, point len) {
	int mm, cbox;
	int j = blockIdx.x*blockDim.x + threadIdx.x + 1;
	int k = blockIdx.y*blockDim.y + threadIdx.y + 1;
	if(j <= no_of_colloid) {
		no_neigh[j] = 0;
		cbox = 1 + pos_colloid[j].cell(len);
		if(k <= nbox) {
			mm = box_neigh[k][cbox];
			for(int i = 1; i <= fluid_no[mm]; i++) 
				neigh_fl[j][dp[j][k] + i] = box_part[mm][i];
			no_neigh[j] = dp[j][nbox + 1];
		}
	}
}
__global__ void sieve(int no_of_colloid, int nbox, int *fluid_no, int **box_neigh, int **dp, point *pos_colloid, point len) {
	int mm, cbox;
	int j = blockDim.x*blockIdx.x + threadIdx.x + 1;
	if(j <= no_of_colloid) {
		cbox = 1 + pos_colloid[j].cell(len);
		dp[j][0] = dp[j][1] = 0;
		for(int k = 2; k <= nbox + 1; k++) {
			mm = box_neigh[k - 1][cbox];
			dp[j][k] = dp[j][k-1] + fluid_no[mm];
		}
	}
}
void neighbour_list_mpcd() {
	blk = dim3((no_of_fluid + thr.x - 1)/thr.x);
	hipMemset(fluid_no, 0, sizeof(int)*(len.prod() + 2));
	d_boxpart<<<blk, thr>>>(box_part, fluid_no, no_of_fluid, pos_fl, len);
	blk = dim3((no_of_colloid + thr.x - 1)/thr.x);
	sieve<<<blk, thr>>>(no_of_colloid, nbox, fluid_no, box_neigh, dp, pos_colloid, len);
	blk = dim3((no_of_colloid + thrs.x - 1)/thrs.x, (nbox + thrs.y - 1)/thrs.y);
	d_neighbour_list_mpcd<<<blk, thrs>>>(box_part, fluid_no, box_neigh, neigh_fl, no_neigh, 
			dp, pos_colloid, pos_fl, no_of_fluid, no_of_colloid, nbox, len);
}
