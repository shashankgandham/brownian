#include "hip/hip_runtime.h"
#include "parameters.cuh"
#include <cstring>
#include <thrust/reduce.h>

hiprandState_t *state;
point *pos_colloid, *pos_fl, *vel_colloid, *vel_fl, *ang_vel_colloid, *f, *ra, *old_force, len = point(30, 30, 30), *cell_vel, **rot, *dump_vel_fl, **u, **vc, **om;
int n = 10, niter = 21000, file = 0, nbin = 300, maxpart = 100, no_of_colloid = 1, nbox, **nbr, **up_nbr, *cnt, *up_cnt, *fluid_no, *iv, *seed, *iy, **dp;
int no_of_fluid = len.prod()*10, *no_neigh, **neigh_fl, **neighbour, *n_neighbour, **box_neigh, **box_part, **cell_part, nn, ran_c = 0, *idum;

double kbt = 1, kbt1 = 1, ndt = 0.1, dv = 0.1, mass_fl = 1.0, mass_colloid = 654.1, sig_colloid = 5.0, eps = 1.0, v0 = 0;
double dt = ndt/(double)n, sigma = 0.80*sig_colloid, I_colloid = 0.1*mass_colloid*sigma*sigma, *potential_colloid, *rana, *ranb;

void initialize() {
	point **ppointers[]  = {&pos_fl, &vel_fl, &f, &pos_colloid, &vel_colloid, &ang_vel_colloid, &old_force, &ra};
	int   **ipointers[]  = {&fluid_no, &n_neighbour, &no_neigh, &cnt, &up_cnt};
	int isize[]          = {(int)len.prod(), no_of_colloid };
	int psize[]          = {no_of_fluid, no_of_colloid};
	hipMallocManaged(&box_part,  (len.prod() + 2)*sizeof(int *));
	hipMallocManaged(&cell_part, (len.prod() + 2)*sizeof(int *));
	hipMallocManaged(&box_neigh, sizeof(int *)*512);
	hipMallocManaged(&neighbour, sizeof(int *)*256);
	hipMallocManaged(&neigh_fl,  sizeof(int *)*(no_of_colloid + 2));
	hipMallocManaged(&dp,  	  sizeof(int *)*(no_of_colloid + 2));
	hipMallocManaged(&nbr, 	  sizeof(int *)*7005);
	hipMallocManaged(&up_nbr,    sizeof(int *)*7005);
	hipMallocManaged(&u,  		(no_of_colloid + 2)*sizeof(point *));
	hipMallocManaged(&rot, (len.prod() + 2)*sizeof(point *));
	hipMallocManaged(&cell_vel, (len.prod() + 2)*sizeof(point));
	hipMallocManaged(&dump_vel_fl, (no_of_fluid + 2)*sizeof(point));
	hipMallocManaged(&iv, sizeof(int)*64);
	hipMallocManaged(&seed, sizeof(int));
	hipMallocManaged(&idum, sizeof(int));
	hipMallocManaged(&iy, sizeof(int));
	hipMallocManaged(&potential_colloid, sizeof(double));
	hipMallocManaged(&vc, (no_of_colloid + 2)*sizeof(point *));
	hipMallocManaged(&om, (no_of_colloid + 2)*sizeof(point *));
	hipMallocManaged(&rana, sizeof(double)*(len.prod() + 2));
	hipMallocManaged(&ranb, sizeof(double)*(len.prod() + 2));
	hipMallocManaged(&state, sizeof(hiprandState_t)*(no_of_fluid + 2));
	*seed = 77777, *idum = 123456789, *iy = 0;
	for(int i = 0; i < 64; i++) iv[i] = 0;
	for(int i = 0; i < 8; i++) {
		if(i < 5)  hipMallocManaged(ipointers[i], (isize[i>0] + 2)*sizeof(int));
		hipMallocManaged(ppointers[i], (psize[i>1] + 2)*sizeof(point));
	}
	for(int i = 0; i <= len.prod(); i++) {
		if(i <= 500)       hipMallocManaged(&box_neigh[i], sizeof(int)*(len.prod()    + 2));
		if(i <= 200)       hipMallocManaged(&neighbour[i], sizeof(int)*(no_of_colloid + 2));
		if(i <= 7000)      hipMallocManaged(&nbr[i],       sizeof(int)*(no_of_colloid + 2));
		if(i <= 7000)      hipMallocManaged(&up_nbr[i],    sizeof(int)*(no_of_colloid + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&neigh_fl[i],  sizeof(int)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&vc[i],  sizeof(point)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&u[i],  sizeof(point)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&om[i],  sizeof(point)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&dp[i],  sizeof(int)*(512)); //512 = nbox
							hipMallocManaged(&box_part[i],  sizeof(int)*(maxpart    + 2));
						   hipMallocManaged(&cell_part[i], sizeof(int)*(maxpart    + 2));
						   hipMallocManaged(&rot[i],		sizeof(point)*4);
	}
}
__global__ void conserv_mom(point *vel, point avr, int no) {
	int i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	if(i <= no) vel[i] = vel[i] - avr;

}
struct add_point: public thrust::binary_function<point &, point &, point &> {
	CUDA_CALLABLE_MEMBER point operator()(const point &a, const point &b) {
		return point(a.x+b.x, a.y+b.y, a.z+b.z);
	}
};

__global__ void initialize_colloid(point *pos_colloid, point *vel_colloid, point *ang_vel_colloid, int no_of_colloid, double sig_colloid, double kbt, double I_colloid, double kbt1, double mass_colloid, point len, hiprandState_t *state) {
	int counter = 0, check;
	double space_limit = 1.3*sig_colloid, ang_vscale_colloid = sqrt(12.0*kbt1/I_colloid), vscale_colloid = sqrt(12.0*kbt1/mass_colloid);
	point avr_vel = point(0, 0, 0), t, temp;
	while(counter < no_of_colloid) {
		t = t.rand(&state[1])*len;
		check = 1;
		for(int j = 1; j <= counter; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < space_limit)? 0: check;
		}
		if(check) pos_colloid[++counter] = t;
	}
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = (vel_colloid[j].rand(&state[1]) - point(0.5, 0.5, 0.5))*vscale_colloid;
		avr_vel += vel_colloid[j];
	}
	avr_vel = avr_vel/no_of_colloid;
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = vel_colloid[j] - avr_vel;
		ang_vel_colloid[j] = (t.rand(&state[1]) - point(0.5, 0.5, 0.5))*ang_vscale_colloid;
	}
}
void initialize_colloid() {
	initialize_colloid<<<1, 1>>>(pos_colloid, vel_colloid, ang_vel_colloid, no_of_colloid, sig_colloid, kbt, I_colloid, kbt1, mass_colloid, len, state);
}
__global__ void d_initialize_fluid(point *pos_fl, point *vel_fl, point *pos_colloid, int no_of_colloid, int no_of_fluid, double kbt, double mass_fl, double sigma, point len, hiprandState_t *state) {
	int check = 1, i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	double vscale_fluid = sqrt(12.0*kbt/mass_fl);
	point t, temp;
	while(true) {
		t = t.rand(&state[i])*len;
		check = 1;
		for(int j = 1; j <= no_of_colloid; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < sigma*0.5)? 0: check;
		}
		if(check) {
			pos_fl[i] = t;
			break;
		}
	}
	vel_fl[i] = (vel_fl[i].rand(&state[i]) - point(0.5, 0.5, 0.5))*vscale_fluid;
}

void initialize_fluid() {
	int thr = 512, blk = (no_of_fluid + thr - 1)/thr;
	point avr_vel;	
	d_initialize_fluid<<<blk, thr>>>(pos_fl, vel_fl, pos_colloid, no_of_colloid, no_of_fluid, kbt, mass_fl, sigma, len, state);
	hipDeviceSynchronize();
	avr_vel = thrust::reduce(vel_fl + 1, vel_fl + no_of_fluid + 1, point(0, 0, 0), add_point());
	avr_vel = avr_vel/no_of_fluid;
	conserv_mom<<<blk, thr>>>(vel_fl, avr_vel, no_of_fluid);
}


__global__ void curand_setup(hiprandState_t *state, int seed) {
	int i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	hiprand_init(seed, i, 0, &state[i]);
}
void initialize_rand() {
	int thr = 512, blk = (no_of_fluid + thr - 1)/thr;
	curand_setup<<<blk, thr>>>(state, *seed);
}
