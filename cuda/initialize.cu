#include "hip/hip_runtime.h"
#include "parameters.cuh"

hiprandState_t *state;
point *pos_colloid, *pos_fl, *vel_colloid, *vel_fl, *ang_vel_colloid, *f, *ra, *old_force, len = point(30, 30, 30), *cell_vel, **rot, *dump_vel_fl, **u, **vc, **om;
int n = 10, niter = 21000, file = 0, nbin = 300, maxpart = 100, no_of_colloid = 10, nbox, **nbr, **up_nbr, *cnt, *up_cnt, *fluid_no, **dp;
int no_of_fluid = len.prod()*10, *no_neigh, **neigh_fl, **neighbour, *n_neighbour, **box_neigh, **box_part, **cell_part, nn, seed = 77777;

double kbt = 1, kbt1 = 1, ndt = 0.1, dv = 0.1, mass_fl = 1.0, mass_colloid = 654.1, sig_colloid = 5.0, eps = 1.0, v0 = 0;
double dt = ndt/(double)n, sigma = 0.80*sig_colloid, I_colloid = 0.1*mass_colloid*sigma*sigma, *potential_colloid, *rana, *ranb;

dim3 thr(512), thrs(32, 32), blk;
void initialize() {
	point **ppointers[]  = {&pos_fl, &vel_fl, &f, &pos_colloid, &vel_colloid, &ang_vel_colloid, &old_force, &ra};
	int   **ipointers[]  = {&fluid_no, &n_neighbour, &no_neigh, &cnt, &up_cnt};
	int isize[]          = {(int)len.prod(), no_of_colloid };
	int psize[]          = {no_of_fluid, no_of_colloid};
	hipMallocManaged(&box_part,  sizeof(int *)*(len.prod() + 2));
	hipMallocManaged(&cell_part, sizeof(int *)*(len.prod() + 2));
	hipMallocManaged(&box_neigh, sizeof(int *)*512);
	hipMallocManaged(&neighbour, sizeof(int *)*256);
	hipMallocManaged(&neigh_fl,  sizeof(int *)*(no_of_colloid + 2));
	hipMallocManaged(&dp,  	  sizeof(int *)*(no_of_colloid + 2));
	hipMallocManaged(&nbr, 	  sizeof(int *)*7005);
	hipMallocManaged(&up_nbr,    sizeof(int *)*7005);
	hipMallocManaged(&u,  		  sizeof(point *)*(no_of_colloid + 2));
	hipMallocManaged(&rot, 	  sizeof(point *)*(len.prod() + 2));
	hipMallocManaged(&cell_vel, (len.prod() + 2)*sizeof(point));
	hipMallocManaged(&dump_vel_fl, (no_of_fluid + 2)*sizeof(point));
	hipMallocManaged(&potential_colloid, sizeof(double));
	hipMallocManaged(&vc, (no_of_colloid + 2)*sizeof(point *));
	hipMallocManaged(&om, (no_of_colloid + 2)*sizeof(point *));
	hipMallocManaged(&state, sizeof(hiprandState_t)*(no_of_fluid + 2));
	for(int i = 0; i < 8; i++) {
		if(i < 5)  hipMallocManaged(ipointers[i], (isize[i>0] + 2)*sizeof(int));
				   hipMallocManaged(ppointers[i], (psize[i>1] + 2)*sizeof(point));
	}
	for(int i = 0; i <= len.prod(); i++) {
		if(i <= 500)       hipMallocManaged(&box_neigh[i], sizeof(int)*(len.prod()    + 2));
		if(i <= 200)       hipMallocManaged(&neighbour[i], sizeof(int)*(no_of_colloid + 2));
		if(i <= 7000)      hipMallocManaged(&nbr[i],       sizeof(int)*(no_of_colloid + 2));
		if(i <= 7000)      hipMallocManaged(&up_nbr[i],    sizeof(int)*(no_of_colloid + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&neigh_fl[i],  sizeof(int)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&vc[i],  sizeof(point)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&u[i],  sizeof(point)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&om[i],  sizeof(point)*(10000 + 2));
		if(i <= no_of_colloid)	hipMallocManaged(&dp[i],  sizeof(int)*(512)); //512 = nbox
								hipMallocManaged(&box_part[i],  sizeof(int)*(maxpart    + 2));
						   		hipMallocManaged(&cell_part[i], sizeof(int)*(maxpart    + 2));
						   		hipMallocManaged(&rot[i],		sizeof(point)*4);
	}
}
__global__ void conserv_mom(point *vel, point avr, int no) {
	int i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	if(i <= no) vel[i] = vel[i] - avr;
}

__global__ void initialize_colloid(point *pos_colloid, point *vel_colloid, point *ang_vel_colloid, int no_of_colloid, double sig_colloid, double kbt, double I_colloid, double kbt1, double mass_colloid, point len, hiprandState_t *state) {
	int counter = 0, check;
	double space_limit = 1.3*sig_colloid, ang_vscale_colloid = sqrt(12.0*kbt1/I_colloid), vscale_colloid = sqrt(12.0*kbt1/mass_colloid);
	point avr_vel = point(0, 0, 0), t, temp;
	while(counter < no_of_colloid) {
		t = t.rand(&state[1])*len;
		check = 1;
		for(int j = 1; j <= counter; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < space_limit)? 0: check;
		}
		if(check) pos_colloid[++counter] = t;
	}
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = (vel_colloid[j].rand(&state[1]) - point(0.5, 0.5, 0.5))*vscale_colloid;
		avr_vel += vel_colloid[j];
	}
	avr_vel = avr_vel/no_of_colloid;
	for(int j = 1; j <= no_of_colloid; j++) {
		vel_colloid[j] = vel_colloid[j] - avr_vel;
		ang_vel_colloid[j] = (t.rand(&state[1]) - point(0.5, 0.5, 0.5))*ang_vscale_colloid;
	}
}
void initialize_colloid() {
	initialize_colloid<<<1, 1>>>(pos_colloid, vel_colloid, ang_vel_colloid, no_of_colloid, sig_colloid, kbt, I_colloid, kbt1, mass_colloid, len, state);
}
__global__ void d_initialize_fluid(point *pos_fl, point *vel_fl, point *pos_colloid, int no_of_colloid, int no_of_fluid, double kbt, double mass_fl, double sigma, point len, hiprandState_t *state) {
	int check = 1, i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	double vscale_fluid = sqrt(12.0*kbt/mass_fl);
	point t, temp;
	while(true) {
		t = t.rand(&state[i])*len;
		check = 1;
		for(int j = 1; j <= no_of_colloid; j++) {
			temp = img(t - pos_colloid[j], len);
			check = (sqrt((temp*temp).sum()) < sigma*0.5)? 0: check;
		}
		if(check) {
			pos_fl[i] = t;
			break;
		}
	}
	vel_fl[i] = (vel_fl[i].rand(&state[i]) - point(0.5, 0.5, 0.5))*vscale_fluid;
}

void initialize_fluid() {
	blk = dim3((no_of_fluid + thr.x - 1)/thr.x);
	point avr_vel;	
	d_initialize_fluid<<<blk, thr>>>(pos_fl, vel_fl, pos_colloid, no_of_colloid, no_of_fluid, kbt, mass_fl, sigma, len, state);
	hipDeviceSynchronize();
	avr_vel = thrust::reduce(thrust::device, vel_fl + 1, vel_fl + no_of_fluid + 1, point(0, 0, 0), add_point())/no_of_fluid;
	conserv_mom<<<blk, thr>>>(vel_fl, avr_vel, no_of_fluid);
}

__global__ void curand_setup(hiprandState_t *state, int seed) {
	int i = blockDim.x*blockIdx.x + threadIdx.x + 1;
	hiprand_init(seed, i, 0, &state[i]);
}
void initialize_rand() {
	blk = dim3((no_of_fluid + thr.x - 1)/thr.x);
	curand_setup<<<blk, thr>>>(state, seed);
}
