#include "hip/hip_runtime.h"
#include "parameters.cuh"

__global__ void imemset(int *x, int n) {
	int i = blockIdx.x*blockDim.x + threadIdx.x + 1;
	if(i <= n) 
		x[i] = 0;
}

int main() {
	clock_t begin = clock();
	point mom;
	double ke_fluid, energy_colloid;
	initialize();
	initialize_rand();
	initialize_colloid();
	initialize_fluid();
	create_box();
	neighbour_list_mpcd();
	neighbour_list_md();
	compute_force_md();
	tumble();
	printf(" After Tumble\n");
	for(nn = 1; nn <= niter; nn++) {
		printf("%12d\n", nn);
		rotation_mpcd();
		run();
		for(int l = 1; l <= n; l++) {
			update_pos_md();
			neighbour_list_md();
			update_pos_mpcd();
			neighbour_list_mpcd();
			if(!(l%10) && nn > 10000) updown_velocity();
			fluid_colloid_collision();
			update_activity_direction();
			compute_force_md();
			update_velocity_colloid();
		}
		hipDeviceSynchronize();
		energy_colloid = *potential_colloid;
		energy_colloid += 0.5*mass_colloid*thrust::transform_reduce(thrust::device, vel_colloid + 1, vel_colloid + no_of_colloid + 1, mod_value(), (double)0, add_double());
		energy_colloid += 0.5*I_colloid*thrust::transform_reduce(thrust::device, ang_vel_colloid + 1, ang_vel_colloid + no_of_colloid + 1, mod_value(), (double)0, add_double());
		mom = thrust::reduce(thrust::device, vel_colloid + 1, vel_colloid + no_of_colloid + 1, point(0, 0, 0), add_point())*mass_colloid;
		mom += thrust::reduce(thrust::device, vel_fl + 1, vel_fl + no_of_fluid + 1, point(0, 0, 0), add_point())*mass_fl;
		ke_fluid = 0.5*mass_fl*thrust::transform_reduce(thrust::device, vel_fl + 1, vel_fl + no_of_fluid + 1, mod_value(), (double)0, add_double());
		printf("%.32lf\n", (mom*mom).sum());
		printf("%.32lf\n", energy_colloid);		
	}
	clock_t end = clock();
	printf("%lf\n", (double)(end - begin)/CLOCKS_PER_SEC);
	return 0;
}
